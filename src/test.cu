#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#define Block_Size 256
#define NUM_BLOCKS_MAX 2147483647

extern "C" void helloworldcuda_ (){
    printf("Hello world CUDA Routine !!\n");
    int deviceCount;
    int dev;
    hipGetDeviceCount(&deviceCount);
    dev=0;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    printf("\nDevice %d: on %d \"%s\"\n", dev, deviceCount, deviceProp.name);
}
__global__ void kernel_aplus(double *a, double *b ,int N){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;//j
    if(tid<N){
        if(tid==0){
            printf("67876\n");
        }
        for(int i=0;i<N;i++){
            a[tid*N+i]=b[tid*N+i]+b[i*N+tid];
        }
    }
    // for(int i=0;i<norb;i++){
    //     for(int j=0;j<norb;j++){
    //         // a[j][i]=F[j][i]+F[i][j]
    //         a[j*N+i]=b[j*N+i]+b[i*N+j]
    //     }
    // }
}
extern "C" void aplus_c_(double *a,double *b,int *NP){   
    printf("Enter GPU aplus  \n");
    int N=*NP;
    int size=N*N;
    int threads_per_block=Block_Size;
    int num_blocks=(N+threads_per_block-1)/threads_per_block;
    double *a_d,*b_d;
    hipMalloc( (void **)&a_d, sizeof(double) * size );
    hipMalloc( (void **)&b_d, sizeof(double) * size );

    hipMemcpy( a_d, a, sizeof(double) * size, hipMemcpyHostToDevice );
    hipMemcpy( b_d, b, sizeof(double) * size, hipMemcpyHostToDevice );
    
    kernel_aplus<<<num_blocks,threads_per_block>>>(a_d,b_d,N);
    hipDeviceSynchronize();
    hipMemcpy( a, a_d, sizeof(double) * size, hipMemcpyDeviceToHost );
    
    hipFree(a_d);
    hipFree(b_d);
}

__device__ __host__ void mat_const12(double *GM1,int index_x,int index_y,double* T,double Dnum,int norb){
    //T 是一个对称矩阵
    int index_xy=index_y*norb+index_x;
    int stride3=norb*norb*norb;
    int stride2=norb*norb;
    for(int i=0;i<norb;i++){
        for(int j=0;j<norb;j++){
            // int uu=j*stride3+i*stride2+index_xy;
            // printf("uu: %d,i: %d,j: %d\n",uu,i,j);
            GM1[j*stride3+i*stride2+index_xy]=T[j*norb+i]*Dnum*2.0;
            // GM1(i0+ii,j0+jj,:,:)=T*D(ix+ii,jx+jj)*2.0d0
            // printf("T[i*norb+j]:%lf  ,  T[i*norb+j]:%lf \n",T[i*norb+j],T[j*norb+i]);
        }
    }

}
__global__ void kernel_dhgen(int *occ,int *tot,double *GM1,double *T,double *D,int nsub,int nact,int norb){
    int i = blockIdx.x * blockDim.x + threadIdx.x;//j
    int ioffset=0;
    int ioffset1=0;
    int i0,i1,i2,ix,joffset,joffset1,j0,j1,j2,jx;
    if(i<nsub){
        for(int ii=0;ii<occ[i];ii++){
            i0=ioffset;
            i1=tot[i];
            i2=occ[i];
            ix=ioffset1;
            joffset=0;
            joffset1=0;
            for(int j=0;j<nsub;j++){
                for(int jj=0;jj<occ[j];jj++){
                    j0=joffset;
                    j1=tot[j];
                    j2=occ[j];
                    jx=joffset1;
                    // GM1(i0+ii,j0+jj,:,:)=T*D(ix+ii,jx+jj)*2.0d0
                    mat_const12(GM1,(i0+ii),(j0+jj),T,D[(jx+jj)*nact+(ix+ii)],norb);
                }
            joffset=joffset+tot[i];
            joffset1=joffset1+occ[i];
            }
        }
        ioffset=ioffset+tot[i];
        ioffset1=ioffset1+occ[i];
    }
}
extern "C" void matgendh_(int *occ,int *tot,double *GM1,double *T,double *D,int *NP,int *NP2,int *NP3){
    printf("Enter GPU dhgen  \n");
    int nsub=*NP;
    int nact=*NP2;
    int norb=*NP3;
    int stride4=norb*norb*norb*norb;
    int stride2=norb*norb;
    int ioffset=0;
    int ioffset1=0;
    int i0,i1,i2,ix,joffset,joffset1,j0,j1,j2,jx=0;
    int threads_per_block=Block_Size;
    int num_blocks=(nsub+threads_per_block-1)/threads_per_block;
    int *d_occ,*d_tot;
    double *d_GM1,*d_T,*d_D;
    hipMalloc( (void **)&d_occ, sizeof(int) * nsub );
    hipMalloc( (void **)&d_tot, sizeof(int) * nsub );
    hipMalloc( (void **)&d_GM1, sizeof(double) * stride4 );
    hipMalloc( (void **)&d_T, sizeof(double) * stride2 );
    hipMalloc( (void **)&d_D, sizeof(double) * nact*nact );
    hipMemcpy( d_occ, occ, sizeof(int) * nsub, hipMemcpyHostToDevice );
    hipMemcpy( d_tot, tot, sizeof(int) * nsub, hipMemcpyHostToDevice );
    hipMemcpy( d_GM1, GM1, sizeof(double) * stride4, hipMemcpyHostToDevice );
    hipMemcpy( d_T, T, sizeof(double) * stride2, hipMemcpyHostToDevice );
    hipMemcpy( d_D, D, sizeof(double) * nact*nact, hipMemcpyHostToDevice );
    kernel_dhgen<<<num_blocks,threads_per_block>>>(d_occ,d_tot,d_GM1,d_T,d_D,nsub,nact,norb);
    hipDeviceSynchronize();
    hipMemcpy( occ, d_occ, sizeof(int) * nsub, hipMemcpyDeviceToHost );
    hipMemcpy( tot, d_tot, sizeof(int) * nsub, hipMemcpyDeviceToHost );
    hipMemcpy( GM1, d_GM1, sizeof(double) * stride4, hipMemcpyDeviceToHost );
    hipMemcpy( T, d_T, sizeof(double) * stride2, hipMemcpyDeviceToHost );
    hipMemcpy( D, d_D, sizeof(double) * nact*nact, hipMemcpyDeviceToHost );
    hipFree(d_occ);
    hipFree(d_tot);
    hipFree(d_GM1);
    hipFree(d_T);
    hipFree(d_D);
}
extern "C" void matgendhcpu_(int *occ,int *tot,double *GM1,double *T,double *D,int *NP,int *NP2,int *NP3){
    int nsub=*NP;
    int nact=*NP2;
    int norb=*NP3;
    int ioffset=0;
    int ioffset1=0;
    int i0,i1,i2,ix,joffset,joffset1,j0,j1,j2,jx=0;
    for(int i=0;i<nsub;i++){
        for(int ii=0;ii<occ[i];ii++){
            i0=ioffset;
            i1=tot[i];
            i2=occ[i];
            ix=ioffset1;
            joffset=0;
            joffset1=0;
            for(int j=0;j<nsub;j++){
                for(int jj=0;jj<occ[j];jj++){
                    j0=joffset;
                    j1=tot[j];
                    j2=occ[j];
                    jx=joffset1;
                    // GM1(i0+ii,j0+jj,:,:)=T*D(ix+ii,jx+jj)*2.0d0
                    mat_const12(GM1,(i0+ii),(j0+jj),T,D[(jx+jj)*nact+(ix+ii)],norb);
                }
                joffset=joffset+tot[i];
                joffset1=joffset1+occ[i];
            }
        }
        ioffset=ioffset+tot[i];
        ioffset1=ioffset1+occ[i];
    }
}
void mat_TM1(double *TM1,double *U,double dtmp,int z,int q,int norb){
    int index_xy=z*norb*norb*norb+q*norb*norb;
    for(int i=0;i<norb;i++){
        for(int j=0;j<norb;j++){
            TM1[i*norb+j]+=U[index_xy+i*norb+j];
            // TM1=TM1+U(:,:,k0+kk,l0+ll)*dtmp
        }
    }
    
}
extern "C" void matgeny_(int *occ,int *tot,double *GM1,double *T,double *D,int *NP,int *NP2,
int *NP3,int *group,double *P,double *U){
    int nsub=*NP;
    int nact=*NP2;
    int norb=*NP3;
    int ioffset=0;
    int ioffset1=0;
    int i0,i1,i2,ix,joffset,joffset1,j0,j1,j2,jx,lx,l0,l1,l2,loffset,loffset1,kx,k0,k1,k2;
    int koffset,koffset1;   
    double dtmp;
    double *TM1,*TM2,*TM3;
    for(int i=0;i<nsub;i++){
        for(int ii=0;ii<occ[i];ii++){
            i0=ioffset;
            i1=tot[i];
            i2=occ[i];
            ix=ioffset1;
            joffset=0;
            joffset1=0;
            for(int j=0;j<nsub;j++){
                for(int jj=0;jj<occ[j];jj++){
                    j0=joffset;
                    j1=tot[j];
                    j2=occ[j];
                    jx=joffset1;

                    memset(TM1, 0, norb*norb * sizeof(double));
                    memset(TM2, 0, norb*norb * sizeof(double));
                    memset(TM3, 0, norb*norb * sizeof(double));

                    koffset=0;
                    koffset1=0;
                    for(int k=0;k<nsub;k++){
                        for(int kk=0;kk<occ[k];kk++){
                            k0=koffset;
                            k1=tot[k];
                            k2=occ[k];
                            kx=koffset1;

                            loffset=0;
                            loffset1=0;
                            for(int l=0;l<nsub;l++){
                                if(group[8*i+j]==group[8*k+l]){
                                    for(int ll=1;ll<occ[l];ll++){
                                        l0=loffset;
                                        l1=tot[l];
                                        l2=occ[l];
                                        lx=loffset1;
                                        dtmp=P[(ix+ii)*nact*nact*nact+(kx+kk)*nact*nact+(lx+ll)*nact+(jx+jj)];
                                        // TM1=TM1+U(:,:,k0+kk,l0+ll)*dtmp
                                    }
                                }
                                if(group[8*i+k]==group[8*j+l]){
                                    for(int ll=1;ll<occ[l];ll++){
                                        l0=loffset;
                                        l1=tot[l];
                                        l2=occ[l];
                                        lx=loffset1;
                                        dtmp=P[(ix+ii)*nact*nact*nact+(jx+jj)*nact*nact+(lx+ll)*nact+(kx+kk)]+
                                             P[(ix+ii)*nact*nact*nact+(lx+ll)*nact*nact+(jx+jj)*nact+(kx+kk)];
                                
                                        // TM2=TM2+U(:,k0+kk,l0+ll,:)*dtmp
                                    }
                                }
                                loffset=loffset+tot[l];
                                loffset1=loffset1+occ[l];
                            }
                        }
                        koffset=koffset+tot[k];
                        koffset1=koffset1+occ[k];
                    }
                    // GM1(i0+ii,:,j0+jj,:)=(TM1+TM2)*2.0d0
                    free(TM1);
                    free(TM2);
                    free(TM3);
                }
                joffset=joffset+tot[j];
                joffset1=joffset1+occ[j];
            }
        }
        ioffset=ioffset+tot[i];
        ioffset1=ioffset1+occ[i];
    }    
}

__global__ void spmv_cusp(const unsigned int M,
                          const double *Ax,
                          const int *Ap,
                          const int *Aj,
                          const double *x,
                          double *y)
{
  const size_t VECTORS_PER_BLOCK = THREADS_PER_BLOCK / THREADS_PER_VECTOR;
  __shared__ volatile double sdata[VECTORS_PER_BLOCK * THREADS_PER_VECTOR + THREADS_PER_VECTOR / 2];
  __shared__ volatile int ptrs[VECTORS_PER_BLOCK][2];

  const int thread_id = THREADS_PER_BLOCK * blockIdx.x + threadIdx.x;  // global thread index
  const int thread_lane = threadIdx.x & (THREADS_PER_VECTOR - 1);      // thread index within the vector
  const int vector_id = thread_id / THREADS_PER_VECTOR;                // global vector index
  const int vector_lane = threadIdx.x / THREADS_PER_VECTOR;            // vector index within the block
  const int num_vectors = VECTORS_PER_BLOCK * gridDim.x;               // total number of active vectors

  for (int row = vector_id; row < M; row += num_vectors) {
    // use two threads to fetch Ap[row] and Ap[row+1],
    // considerably faster than the straightforward version
    if (thread_lane < 2) {
      ptrs[vector_lane][thread_lane] = Ap[row + thread_lane];
    }
    const int row_start = ptrs[vector_lane][0];  // same as row+start = Ap[row]
    const int row_end = ptrs[vector_lane][1];    // same as row_end = Ap[row+1]

    // initialize local sum
    double sum = 0.0;

    if (THREADS_PER_VECTOR == 32 && row_end - row_start > 32) {
      int jj = row_start - (row_start & (THREADS_PER_VECTOR - 1)) + thread_lane;
      if (jj >= row_start && jj < row_end) sum += Ax[jj] * x[Aj[jj]];
      for (jj += THREADS_PER_VECTOR; jj < row_end; jj += THREADS_PER_VECTOR) sum += Ax[jj] * x[Aj[jj]];
    } else {
      for (int jj = row_start + thread_lane; jj < row_end; jj += THREADS_PER_VECTOR) sum += Ax[jj] * x[Aj[jj]];
    }
    // Store local sum in the shared memory
    sdata[threadIdx.x] = sum;
    // Reduce local sums to row sum
    double tmp;
    if (THREADS_PER_VECTOR > 16) {
      tmp = sdata[threadIdx.x + 16];
      sum += tmp;
      sdata[threadIdx.x] = sum;
    }
    if (THREADS_PER_VECTOR > 8) {
      tmp = sdata[threadIdx.x + 8];
      sum += tmp;
      sdata[threadIdx.x] = sum;
    }
    if (THREADS_PER_VECTOR > 4) {
      tmp = sdata[threadIdx.x + 4];
      sum += tmp;
      sdata[threadIdx.x] = sum;
    }
    if (THREADS_PER_VECTOR > 2) {
      tmp = sdata[threadIdx.x + 2];
      sum += tmp;
      sdata[threadIdx.x] = sum;
    }
    if (THREADS_PER_VECTOR > 1) {
      tmp = sdata[threadIdx.x + 1];
      sum += tmp;
      sdata[threadIdx.x] = sum;
    }
    // First thread writes the result
    if (thread_lane == 0) {y[row] = sdata[threadIdx.x];}
  }
}

extern "C"  void xjf_csr_(int *Anrowsc,
                                int *Annzc,
                                const double *Avals,
                                const int *Arows,
                                const int *Acols,
                                const double *xd,
                                double *b2)
{
  //定义A device
  int Anrows=*Anrowsc;
  int Annz=*Annzc;
  int *Arow_offset,*Aclo;
  double *Avalue,*xD,*y;

  hipMalloc(((void **)(&xD)),Anrows* sizeof(double ));//
  hipMemcpy(xD,xd,Anrows* sizeof(double ),hipMemcpyHostToDevice);

  hipMalloc(((void **)(&Avalue)),Annz* sizeof(double ));//
  hipMemcpy(Avalue,Avals,Annz* sizeof(double ),hipMemcpyHostToDevice);

  hipMalloc(((void **)(&Arow_offset)),(Anrows+1)* sizeof(int ));//
  hipMemcpy(Arow_offset,Arows,(Anrows+1)* sizeof(int ),hipMemcpyHostToDevice);

  hipMalloc(((void **)(&Aclo)),Annz* sizeof(int ));
  hipMemcpy(Aclo,Acols,Annz* sizeof(int ),hipMemcpyHostToDevice);//A.cols

  hipMalloc(((void **)(&y)),Anrows* sizeof(double ));
//   hipMemcpy(y,b2,Anrows* sizeof(double),hipMemcpyHostToDevice);//y

  const size_t VECTORS_PER_BLOCK  = THREADS_PER_BLOCK / THREADS_PER_VECTOR;//一个块中计算了多少行
  const size_t MAX_BLOCKS  = 2048;//cusp::system::cuda::detail::max_active_blocks
  const size_t NUM_BLOCKS = min(MAX_BLOCKS, (Anrows + (VECTORS_PER_BLOCK - 1)) / VECTORS_PER_BLOCK);
  
  spmv_cusp<<< NUM_BLOCKS,THREADS_PER_BLOCK,0 >>>(Anrows,Avalue,Arow_offset,Aclo,xD,y);
  hipDeviceSynchronize();
  hipMemcpy(b2,y,Anrows* sizeof(double ),hipMemcpyDeviceToHost);
  
  hipFree(Arow_offset);
  hipFree(Aclo);
  hipFree(Avalue);
  hipFree(xD);
  hipFree(y);
}


__global__ void kernel_dhgen(int *occ,int *tot,double *GM1,
                             double *T,double *D,int nsub,
                             int nact,int norb,int i){
    int ii = blockIdx.x * blockDim.x + threadIdx.x;//j
    int ioffset=0;
    int ioffset1=0;
    int i0,ix,joffset,joffset1,j0,jx;
    if(ii<occ[i]){
        i0=ioffset;
        ix=ioffset1;
        joffset=0;
        joffset1=0;
        for(int j=0;j<nsub;j++){
            for(int jj=0;jj<occ[j];jj++){
                j0=joffset;
                jx=joffset1;
                // GM1(i0+ii,j0+jj,:,:)=T*D(ix+ii,jx+jj)*2.0d0
                mat_const12(GM1,(i0+ii),(j0+jj),T,D[(jx+jj)*nact+(ix+ii)],norb);
            }
            joffset=joffset+tot[i];
            joffset1=joffset1+occ[i];
        }
    }
    ioffset=ioffset+tot[i];
    ioffset1=ioffset1+occ[i];
}

extern "C" void matgendh_(int *occ,int *tot,double *GM1,
                          double *T,double *D,int *NP,
                          int *NP2,int *NP3){
    printf("Enter GPU dhgen  \n");
    int nsub=*NP;
    int nact=*NP2;
    int norb=*NP3;
    int stride4=norb*norb*norb*norb;
    int stride2=norb*norb;
    int threads_per_block=THREADS_PER_BLOCK;
    int num_blocks=(nsub+threads_per_block-1)/threads_per_block;
    int *d_occ,*d_tot;
    double *d_GM1,*d_T,*d_D;
    hipMalloc( (void **)&d_occ, sizeof(int) * nsub );
    hipMalloc( (void **)&d_tot, sizeof(int) * nsub );
    hipMalloc( (void **)&d_GM1, sizeof(double) * stride4 );
    hipMalloc( (void **)&d_T, sizeof(double) * stride2 );
    hipMalloc( (void **)&d_D, sizeof(double) * nact*nact );
    hipMemcpy( d_occ, occ, sizeof(int) * nsub, hipMemcpyHostToDevice );
    hipMemcpy( d_tot, tot, sizeof(int) * nsub, hipMemcpyHostToDevice );
    hipMemcpy( d_GM1, GM1, sizeof(double) * stride4, hipMemcpyHostToDevice );
    hipMemcpy( d_T, T, sizeof(double) * stride2, hipMemcpyHostToDevice );
    hipMemcpy( d_D, D, sizeof(double) * nact*nact, hipMemcpyHostToDevice );
    for (int  i = 0; i < nsub; i++)
    {
        kernel_dhgen<<<num_blocks,threads_per_block>>>(d_occ,d_tot,d_GM1,d_T,d_D,nsub,nact,norb,i);
    }
    hipDeviceSynchronize();
    hipMemcpy( occ, d_occ, sizeof(int) * nsub, hipMemcpyDeviceToHost );
    hipMemcpy( tot, d_tot, sizeof(int) * nsub, hipMemcpyDeviceToHost );
    hipMemcpy( GM1, d_GM1, sizeof(double) * stride4, hipMemcpyDeviceToHost );
    hipMemcpy( T, d_T, sizeof(double) * stride2, hipMemcpyDeviceToHost );
    hipMemcpy( D, d_D, sizeof(double) * nact*nact, hipMemcpyDeviceToHost );
    hipFree(d_occ);
    hipFree(d_tot);
    hipFree(d_GM1);
    hipFree(d_T);
    hipFree(d_D);
}

extern "C" void xjfserial_host_(int *rows, 
                            int *nz_num,
                            int * Ap, 
                            int * Aj, 
                            double *dddxjf, 
                            double *hfdjshx,    
                            double *y)    
{
    // printf("\nserial_host_\n");
    int num_rows=*rows;
    for (int i = 0; i < num_rows; i++){
        // printf("%lf ",hfdjshx[i]);
        const int row_start = Ap[i]-1;
        const int row_end   = Ap[i+1]-1;
        double sum = 0.0;
        for (int jj = row_start; jj < row_end; jj++) {            
            const int j = Aj[jj]-1;  //column index
            sum += hfdjshx[j] * dddxjf[jj];
        }
        y[i] = sum; 
    }
}
