#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#define Block_Size 256
#define NUM_BLOCKS_MAX 2147483647

extern "C" void helloworldcuda_ (){
    printf("Hello world CUDA Routine !!\n");
    int deviceCount;
    int dev;
    hipGetDeviceCount(&deviceCount);
    dev=0;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    printf("\nDevice %d: on %d \"%s\"\n", dev, deviceCount, deviceProp.name);
}
__global__ void kernel_aplus(double *a, double *b ,int N){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;//j
    if(tid<N){
        if(tid==0){
            printf("67876\n");
        }
        for(int i=0;i<N;i++){
            a[tid*N+i]=b[tid*N+i]+b[i*N+tid];
        }
    }
    // for(int i=0;i<norb;i++){
    //     for(int j=0;j<norb;j++){
    //         // a[j][i]=F[j][i]+F[i][j]
    //         a[j*N+i]=b[j*N+i]+b[i*N+j]
    //     }
    // }
}
extern "C" void aplus_c_(double *a,double *b,int *NP){   
    printf("Enter GPU aplus  \n");
    int N=*NP;
    int size=N*N;
    int threads_per_block=Block_Size;
    int num_blocks=(N+threads_per_block-1)/threads_per_block;
    double *a_d,*b_d;
    hipMalloc( (void **)&a_d, sizeof(double) * size );
    hipMalloc( (void **)&b_d, sizeof(double) * size );

    hipMemcpy( a_d, a, sizeof(double) * size, hipMemcpyHostToDevice );
    hipMemcpy( b_d, b, sizeof(double) * size, hipMemcpyHostToDevice );
    
    kernel_aplus<<<num_blocks,threads_per_block>>>(a_d,b_d,N);
    hipDeviceSynchronize();
    hipMemcpy( a, a_d, sizeof(double) * size, hipMemcpyDeviceToHost );
    
    hipFree(a_d);
    hipFree(b_d);
}

__device__ __host__ void mat_const12(double *GM1,int index_x,int index_y,double* T,double Dnum,int norb){
    //T 是一个对称矩阵
    int index_xy=index_y*norb+index_x;
    int stride3=norb*norb*norb;
    int stride2=norb*norb;
    for(int i=0;i<norb;i++){
        for(int j=0;j<norb;j++){
            // int uu=j*stride3+i*stride2+index_xy;
            // printf("uu: %d,i: %d,j: %d\n",uu,i,j);
            GM1[j*stride3+i*stride2+index_xy]=T[j*norb+i]*Dnum*2.0;
            // GM1(i0+ii,j0+jj,:,:)=T*D(ix+ii,jx+jj)*2.0d0
            // printf("T[i*norb+j]:%lf  ,  T[i*norb+j]:%lf \n",T[i*norb+j],T[j*norb+i]);
        }
    }

}
__global__ void kernel_dhgen(int *occ,int *tot,double *GM1,double *T,double *D,int nsub,int nact,int norb){
    int i = blockIdx.x * blockDim.x + threadIdx.x;//j
    int ioffset=0;
    int ioffset1=0;
    int i0,i1,i2,ix,joffset,joffset1,j0,j1,j2,jx;
    if(i<nsub){
        for(int ii=0;ii<occ[i];ii++){
            i0=ioffset;
            i1=tot[i];
            i2=occ[i];
            ix=ioffset1;
            joffset=0;
            joffset1=0;
            for(int j=0;j<nsub;j++){
                for(int jj=0;jj<occ[j];jj++){
                    j0=joffset;
                    j1=tot[j];
                    j2=occ[j];
                    jx=joffset1;
                    // GM1(i0+ii,j0+jj,:,:)=T*D(ix+ii,jx+jj)*2.0d0
                    mat_const12(GM1,(i0+ii),(j0+jj),T,D[(jx+jj)*nact+(ix+ii)],norb);
                }
            joffset=joffset+tot[i];
            joffset1=joffset1+occ[i];
            }
        }
        ioffset=ioffset+tot[i];
        ioffset1=ioffset1+occ[i];
    }
}
extern "C" void matgendh_(int *occ,int *tot,double *GM1,double *T,double *D,int *NP,int *NP2,int *NP3){
    printf("Enter GPU dhgen  \n");
    int nsub=*NP;
    int nact=*NP2;
    int norb=*NP3;
    int stride4=norb*norb*norb*norb;
    int stride2=norb*norb;
    int ioffset=0;
    int ioffset1=0;
    int i0,i1,i2,ix,joffset,joffset1,j0,j1,j2,jx=0;
    int threads_per_block=Block_Size;
    int num_blocks=(nsub+threads_per_block-1)/threads_per_block;
    int *d_occ,*d_tot;
    double *d_GM1,*d_T,*d_D;
    hipMalloc( (void **)&d_occ, sizeof(int) * nsub );
    hipMalloc( (void **)&d_tot, sizeof(int) * nsub );
    hipMalloc( (void **)&d_GM1, sizeof(double) * stride4 );
    hipMalloc( (void **)&d_T, sizeof(double) * stride2 );
    hipMalloc( (void **)&d_D, sizeof(double) * nact*nact );
    hipMemcpy( d_occ, occ, sizeof(int) * nsub, hipMemcpyHostToDevice );
    hipMemcpy( d_tot, tot, sizeof(int) * nsub, hipMemcpyHostToDevice );
    hipMemcpy( d_GM1, GM1, sizeof(double) * stride4, hipMemcpyHostToDevice );
    hipMemcpy( d_T, T, sizeof(double) * stride2, hipMemcpyHostToDevice );
    hipMemcpy( d_D, D, sizeof(double) * nact*nact, hipMemcpyHostToDevice );
    kernel_dhgen<<<num_blocks,threads_per_block>>>(d_occ,d_tot,d_GM1,d_T,d_D,nsub,nact,norb);
    hipDeviceSynchronize();
    hipMemcpy( occ, d_occ, sizeof(int) * nsub, hipMemcpyDeviceToHost );
    hipMemcpy( tot, d_tot, sizeof(int) * nsub, hipMemcpyDeviceToHost );
    hipMemcpy( GM1, d_GM1, sizeof(double) * stride4, hipMemcpyDeviceToHost );
    hipMemcpy( T, d_T, sizeof(double) * stride2, hipMemcpyDeviceToHost );
    hipMemcpy( D, d_D, sizeof(double) * nact*nact, hipMemcpyDeviceToHost );
    hipFree(d_occ);
    hipFree(d_tot);
    hipFree(d_GM1);
    hipFree(d_T);
    hipFree(d_D);
}
extern "C" void matgendhcpu_(int *occ,int *tot,double *GM1,double *T,double *D,int *NP,int *NP2,int *NP3){
    int nsub=*NP;
    int nact=*NP2;
    int norb=*NP3;
    int ioffset=0;
    int ioffset1=0;
    int i0,i1,i2,ix,joffset,joffset1,j0,j1,j2,jx=0;
    for(int i=0;i<nsub;i++){
        for(int ii=0;ii<occ[i];ii++){
            i0=ioffset;
            i1=tot[i];
            i2=occ[i];
            ix=ioffset1;
            joffset=0;
            joffset1=0;
            for(int j=0;j<nsub;j++){
                for(int jj=0;jj<occ[j];jj++){
                    j0=joffset;
                    j1=tot[j];
                    j2=occ[j];
                    jx=joffset1;
                    // GM1(i0+ii,j0+jj,:,:)=T*D(ix+ii,jx+jj)*2.0d0
                    mat_const12(GM1,(i0+ii),(j0+jj),T,D[(jx+jj)*nact+(ix+ii)],norb);
                }
            joffset=joffset+tot[i];
            joffset1=joffset1+occ[i];
            }
        }
        ioffset=ioffset+tot[i];
        ioffset1=ioffset1+occ[i];
    }
}
void mat_TM1(double *TM1,double *U,double dtmp,int z,int q,int norb){
    int index_xy=z*norb*norb*norb+q*norb*norb;
    for(int i=0;i<norb;i++){
        for(int j=0;j<norb;j++){
            TM1[i*norb+j]+=U[index_xy+i*norb+j];
            // TM1=TM1+U(:,:,k0+kk,l0+ll)*dtmp
        }
    }
    
}
extern "C" void matgeny_(int *occ,int *tot,double *GM1,double *T,double *D,int *NP,int *NP2,
int *NP3,int *group,double *P,double *U){
    int nsub=*NP;
    int nact=*NP2;
    int norb=*NP3;
    int ioffset=0;
    int ioffset1=0;
    int i0,i1,i2,ix,joffset,joffset1,j0,j1,j2,jx,lx,l0,l1,l2,loffset,loffset1,kx,k0,k1,k2;
    int koffset,koffset1;   
    double dtmp;
    double *TM1,*TM2,*TM3;
    for(int i=0;i<nsub;i++){
        for(int ii=0;ii<occ[i];ii++){
            i0=ioffset;
            i1=tot[i];
            i2=occ[i];
            ix=ioffset1;
            joffset=0;
            joffset1=0;
            for(int j=0;j<nsub;j++){
                for(int jj=0;jj<occ[j];jj++){
                    j0=joffset;
                    j1=tot[j];
                    j2=occ[j];
                    jx=joffset1;

                    memset(TM1, 0, norb*norb * sizeof(double));
                    memset(TM2, 0, norb*norb * sizeof(double));
                    memset(TM3, 0, norb*norb * sizeof(double));

                    koffset=0;
                    koffset1=0;
                    for(int k=0;k<nsub;k++){
                        for(int kk=0;kk<occ[k];kk++){
                            k0=koffset;
                            k1=tot[k];
                            k2=occ[k];
                            kx=koffset1;

                            loffset=0;
                            loffset1=0;
                            for(int l=0;l<nsub;l++){
                                if(group[8*i+j]==group[8*k+l]){
                                    for(int ll=1;ll<occ[l];ll++){
                                        l0=loffset;
                                        l1=tot[l];
                                        l2=occ[l];
                                        lx=loffset1;
                                        dtmp=P[(ix+ii)*nact*nact*nact+(kx+kk)*nact*nact+(lx+ll)*nact+(jx+jj)];
                                        // TM1=TM1+U(:,:,k0+kk,l0+ll)*dtmp
                                    }
                                }
                                if(group[8*i+k]==group[8*j+l]){
                                    for(int ll=1;ll<occ[l];ll++){
                                        l0=loffset;
                                        l1=tot[l];
                                        l2=occ[l];
                                        lx=loffset1;
                                        dtmp=P[(ix+ii)*nact*nact*nact+(jx+jj)*nact*nact+(lx+ll)*nact+(kx+kk)]+
                                             P[(ix+ii)*nact*nact*nact+(lx+ll)*nact*nact+(jx+jj)*nact+(kx+kk)];
                                
                                        // TM2=TM2+U(:,k0+kk,l0+ll,:)*dtmp
                                    }
                                }
                                loffset=loffset+tot[l];
                                loffset1=loffset1+occ[l];
                            }
                        }
                        koffset=koffset+tot[k];
                        koffset1=koffset1+occ[k];
                    }
                    // GM1(i0+ii,:,j0+jj,:)=(TM1+TM2)*2.0d0
                    free(TM1);
                    free(TM2);
                    free(TM3);
                }
                joffset=joffset+tot[j];
                joffset1=joffset1+occ[j];
            }
        }
        ioffset=ioffset+tot[i];
        ioffset1=ioffset1+occ[i];
    }    
}