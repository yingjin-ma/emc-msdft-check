#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#define THREADS_PER_BLOCK 256 
#define NUM_BLOCKS_MAX 2147483647 //2的31次方-1
#define THREADS_PER_VECTOR 32
#define HESSIAN3_SHARED_MEM_BLOCK_SIZE 6144


__global__ void stretch(double *a,
                        double *b,
                        int N){
    int index= blockIdx.x * blockDim.x + threadIdx.x;
    int totalThreads = blockDim.x * gridDim.x;
    int i,j,stride2=N*N;
    double temp1,temp2;
    for(;index<stride2;index+=totalThreads){
        i=index/N;
        j=index%N;
        temp1=b[index];
        temp2=b[j*N+i];
        a[index]=temp1+temp2;
    }
}

__global__ void kernel_update_hess_stretch(double *aplus_ck,int norb,
                                           double *hess_ck,double *dh_ck,double *Y_ck,
                                           double *Hdiag_c,int stride3,int stride2){
    int index= blockIdx.x * blockDim.x + threadIdx.x;
    int totalThreads = blockDim.x * gridDim.x;
    double adelt=0.0;
    int dij,drj,dis,drs;
    double temp,temp1,temp2;
    // if(index==0){printf("\n674374\n");}
    double dh_ck_rsij,dh_ck_isrj,dh_ck_rjis,dh_ck_ijrs;
    double Y_ck_risj,Y_ck_irsj,Y_ck_rijs,Y_ck_irjs;
    double aplus_ck_rs,aplus_ck_is,aplus_ck_rj,aplus_ck_ij;
    double hess_ck_riri;
    for(;index<norb*stride3;index+=totalThreads){
        int r = index / stride3;
        int i = (index / stride2) % norb;
        int s = (index / norb) % norb;
        int j = index % norb;
        dij = (i == j) ? 1.0 : 0.0;
        drj = (r == j) ? 1.0 : 0.0;
        dis = (i == s) ? 1.0 : 0.0;
        drs = (r == s) ? 1.0 : 0.0;
        adelt=0.0;
        aplus_ck_rs=aplus_ck[r*norb+s];
        aplus_ck_is=aplus_ck[i*norb+s];
        aplus_ck_rj=aplus_ck[r*norb+j];
        aplus_ck_ij=aplus_ck[i*norb+j];
        adelt=aplus_ck_rs*dij-aplus_ck_is*drj-aplus_ck_rj*dis+aplus_ck_ij*drs;
        temp=hess_ck[r*stride3+i*stride2+s*norb+j];
        dh_ck_rsij=dh_ck[r*stride3+s*stride2+i*norb+j];
        dh_ck_isrj=dh_ck[i*stride3+s*stride2+r*norb+j];
        dh_ck_rjis=dh_ck[r*stride3+j*stride2+i*norb+s];
        dh_ck_ijrs=dh_ck[i*stride3+j*stride2+r*norb+s];
        temp1=dh_ck_rsij-dh_ck_isrj-dh_ck_rjis+dh_ck_ijrs;
        Y_ck_risj=Y_ck[r*stride3+i*stride2+s*norb+j];
        Y_ck_irsj=Y_ck[i*stride3+r*stride2+s*norb+j];
        Y_ck_rijs=Y_ck[r*stride3+i*stride2+j*norb+s];
        Y_ck_irjs=Y_ck[i*stride3+r*stride2+j*norb+s];
        temp2=  Y_ck_risj-Y_ck_irsj-Y_ck_rijs+Y_ck_irjs;
        hess_ck[r*stride3+i*stride2+s*norb+j]=temp+temp1+temp2-0.5*adelt;
        if(s==norb&&j==norb){
            hess_ck_riri=hess_ck[r*stride3+i*stride2+r*norb+i];
            Hdiag_c[r*norb+i]=hess_ck_riri;
        }
    }
}

extern "C" void hess3c_(double *aplus_c,double *F_c,int *NP,
                        double *hess_c,double *dh_c,double *Y_c,
                        double *Hdiag_c){
    printf("\nEnter hess3c_ C++\n");
    int norb=*NP;
    int stride4=norb*norb*norb*norb,stride3=norb*norb*norb,stride2=norb*norb;
    int threads_per_block=THREADS_PER_BLOCK,num_blocks=(stride2+threads_per_block-1)/threads_per_block;
    num_blocks=min(NUM_BLOCKS_MAX,num_blocks);
    printf("num_blocks\n  %d ",num_blocks);
    double *aplus_c_d,*F_c_d;
    hipMalloc( (void **)&aplus_c_d, sizeof(double) * stride2 );
    hipMalloc( (void **)&F_c_d, sizeof(double) * stride2 );
    hipMemcpy( aplus_c_d, aplus_c, sizeof(double) * stride2, hipMemcpyHostToDevice );
    hipMemcpy( F_c_d, F_c, sizeof(double) * stride2, hipMemcpyHostToDevice );
    stretch<<<num_blocks,threads_per_block>>>(aplus_c_d,F_c_d,norb);
    hipDeviceSynchronize();
    // for(int index=0;index<stride2;index++){
    //     int i=index/norb;
    //     int j=index%norb;
    //     aplus_c[index]=F_c[index]+F_c[j*norb+i];
    // }
    // for(int i=0;i<norb;i++){
    //     for(int j=0;j<norb;j++){
    //         aplus_c[j*norb+i]=F_c[j*norb+i]+F_c[i*norb+j];
    //     }
    // }
    
    double *hess_c_d,*dh_c_d,*Y_c_d,*Hdiag_c_d;
    hipMalloc( (void **)&hess_c_d, sizeof(double) * stride4 );
    hipMalloc( (void **)&dh_c_d, sizeof(double) * stride4 );
    hipMalloc( (void **)&Y_c_d, sizeof(double) * stride4 );
    hipMalloc( (void **)&Hdiag_c_d, sizeof(double) * stride2 );
    hipMemcpy( hess_c_d, hess_c, sizeof(double) * stride4, hipMemcpyHostToDevice );
    hipMemcpy( dh_c_d, dh_c, sizeof(double) * stride4, hipMemcpyHostToDevice );
    hipMemcpy( Y_c_d, Y_c, sizeof(double) * stride4, hipMemcpyHostToDevice );
    hipMemcpy( Hdiag_c_d, Hdiag_c, sizeof(double) * stride2, hipMemcpyHostToDevice );
    num_blocks=(stride4+threads_per_block-1)/threads_per_block;
    printf("num_block%d\n",num_blocks);
    kernel_update_hess_stretch<<<num_blocks,threads_per_block>>>(aplus_c_d,norb,
                                           hess_c_d,dh_c_d,Y_c_d,Hdiag_c_d,stride3,stride2);
    hipDeviceSynchronize();
    
    hipMemcpy( aplus_c, aplus_c_d, sizeof(double) * stride2, hipMemcpyDeviceToHost );
    hipMemcpy( hess_c, hess_c_d, sizeof(double) * stride4, hipMemcpyDeviceToHost );
    hipMemcpy( Hdiag_c, Hdiag_c_d, sizeof(double) * stride4, hipMemcpyDeviceToHost );

    hipFree(aplus_c_d);
    hipFree(F_c_d);
    hipFree(hess_c_d);
    hipFree(dh_c_d);
    hipFree(Y_c_d);
    hipFree(Hdiag_c_d);

    // double adelt=0.0;
    // int dij=0,drj=0,dis=0,drs=0;
    // for(int index=0;index<norb*stride3;index++){
    //     int r = index / stride3;
    //     int i = (index / stride2) % norb;
    //     int s = (index / norb) % norb;
    //     int j = index % norb;
    //     dij = (i == j) ? 1.0 : 0.0;
    //     drj = (r == j) ? 1.0 : 0.0;
    //     dis = (i == s) ? 1.0 : 0.0;
    //     drs = (r == s) ? 1.0 : 0.0;
    //     adelt=0.0;
    //     adelt=aplus_c[r*norb+s]*dij-aplus_c[i*norb+s]*drj-aplus_c[r*norb+j]*dis+
    //                 aplus_c[i*norb+j]*drs;
        
    //     hess_c[r*stride3+i*stride2+s*norb+j]=
    //     hess_c[r*stride3+i*stride2+s*norb+j]+dh_c[r*stride3+s*stride2+i*norb+j]-
    //     dh_c[i*stride3+s*stride2+r*norb+j]-dh_c[r*stride3+j*stride2+i*norb+s]+
    //     dh_c[i*stride3+j*stride2+r*norb+s];

    //     hess_c[r*stride3+i*stride2+s*norb+j]=hess_c[r*stride3+i*stride2+s*norb+j]+
    //     Y_c[r*stride3+i*stride2+s*norb+j]-Y_c[i*stride3+r*stride2+s*norb+j]-
    //     Y_c[r*stride3+i*stride2+j*norb+s]+Y_c[i*stride3+r*stride2+j*norb+s];

    //     hess_c[r*stride3+i*stride2+s*norb+j]-=0.5*adelt;

    //     if(s==norb&&j==norb){
    //         Hdiag_c[r*norb+i]=hess_c[r*stride3+i*stride2+r*norb+i];
    //     }

    // }
    // for(int r=0;r<norb;r++){
    //     for(int i=0;i<norb;i++){
    //         for(int s=0;s<norb;s++){
    //             for(int j=0;j<norb;j++){
    //                 adelt=0.0;
    //                 dij = (i == j) ? 1.0 : 0.0;
    //                 drj = (r == j) ? 1.0 : 0.0;
    //                 dis = (i == s) ? 1.0 : 0.0;
    //                 drs = (r == s) ? 1.0 : 0.0;
    //                 adelt=aplus_c[r*norb+s]*dij-aplus_c[i*norb+s]*drj-aplus_c[r*norb+j]*dis+
    //                 aplus_c[i*norb+j]*drs;

    //                 hess_c[r*stride3+i*stride2+s*norb+j]=
    //                 hess_c[r*stride3+i*stride2+s*norb+j]+dh_c[r*stride3+s*stride2+i*norb+j]-
    //                 dh_c[i*stride3+s*stride2+r*norb+j]-dh_c[r*stride3+j*stride2+i*norb+s]+
    //                 dh_c[i*stride3+j*stride2+r*norb+s];

    //                 hess_c[r*stride3+i*stride2+s*norb+j]=hess_c[r*stride3+i*stride2+s*norb+j]+
    //                 Y_c[r*stride3+i*stride2+s*norb+j]-Y_c[i*stride3+r*stride2+s*norb+j]-
    //                 Y_c[r*stride3+i*stride2+j*norb+s]+Y_c[i*stride3+r*stride2+j*norb+s];

    //                 hess_c[r*stride3+i*stride2+s*norb+j]-=0.5*adelt;
    //             }
    //         }
    //         Hdiag_c[r*norb+i]=hess_c[r*stride3+i*stride2+r*norb+i];
    //     }
    // }
    printf("End C++\n");
}

__device__ __host__ void mat_const12(double *GM1,int index_x,
                                     int index_y,double* T,
                                     double Dnum,int norb){
    //T 是一个对称矩阵
    int index_xy=index_y*norb+index_x;
    int stride3=norb*norb*norb;
    int stride2=norb*norb;
    for(int i=0;i<norb;i++){
        for(int j=0;j<norb;j++){
            // int uu=j*stride3+i*stride2+index_xy;
            // printf("uu: %d,i: %d,j: %d\n",uu,i,j);
            GM1[j*stride3+i*stride2+index_xy]=T[j*norb+i]*Dnum*2.0;
            // GM1(i0+ii,j0+jj,:,:)=T*D(ix+ii,jx+jj)*2.0d0
            // printf("T[i*norb+j]:%lf  ,  T[i*norb+j]:%lf \n",T[i*norb+j],T[j*norb+i]);
        }
    }

}

__global__ void spmv_cusp(const unsigned int M,
                          const double *Ax,
                          const int *Ap,
                          const int *Aj,
                          const double *x,
                          double *y)
{
  const size_t VECTORS_PER_BLOCK = THREADS_PER_BLOCK / THREADS_PER_VECTOR;
  __shared__ volatile double sdata[VECTORS_PER_BLOCK * THREADS_PER_VECTOR + THREADS_PER_VECTOR / 2];
  __shared__ volatile int ptrs[VECTORS_PER_BLOCK][2];

  const int thread_id = THREADS_PER_BLOCK * blockIdx.x + threadIdx.x;  // global thread index
  const int thread_lane = threadIdx.x & (THREADS_PER_VECTOR - 1);      // thread index within the vector
  const int vector_id = thread_id / THREADS_PER_VECTOR;                // global vector index
  const int vector_lane = threadIdx.x / THREADS_PER_VECTOR;            // vector index within the block
  const int num_vectors = VECTORS_PER_BLOCK * gridDim.x;               // total number of active vectors

  for (int row = vector_id; row < M; row += num_vectors) {
    // use two threads to fetch Ap[row] and Ap[row+1],
    // considerably faster than the straightforward version
    if (thread_lane < 2) {
      ptrs[vector_lane][thread_lane] = Ap[row + thread_lane]-1;
    }
    const int row_start = ptrs[vector_lane][0];  // same as row+start = Ap[row]
    const int row_end = ptrs[vector_lane][1];    // same as row_end = Ap[row+1]

    // initialize local sum
    double sum = 0.0;

    if (THREADS_PER_VECTOR == 32 && row_end - row_start > 32) {
      int jj = row_start - (row_start & (THREADS_PER_VECTOR - 1)) + thread_lane;
      if (jj >= row_start && jj < row_end) sum += Ax[jj] * x[Aj[jj]-1];
      for (jj += THREADS_PER_VECTOR; jj < row_end; jj += THREADS_PER_VECTOR) sum += Ax[jj] * x[Aj[jj]-1];
    } else {
      for (int jj = row_start + thread_lane; jj < row_end; jj += THREADS_PER_VECTOR) sum += Ax[jj] * x[Aj[jj]-1];
    }
    // Store local sum in the shared memory
    sdata[threadIdx.x] = sum;
    // Reduce local sums to row sum
    double tmp;
    if (THREADS_PER_VECTOR > 16) {
      tmp = sdata[threadIdx.x + 16];
      sum += tmp;
      sdata[threadIdx.x] = sum;
    }
    if (THREADS_PER_VECTOR > 8) {
      tmp = sdata[threadIdx.x + 8];
      sum += tmp;
      sdata[threadIdx.x] = sum;
    }
    if (THREADS_PER_VECTOR > 4) {
      tmp = sdata[threadIdx.x + 4];
      sum += tmp;
      sdata[threadIdx.x] = sum;
    }
    if (THREADS_PER_VECTOR > 2) {
      tmp = sdata[threadIdx.x + 2];
      sum += tmp;
      sdata[threadIdx.x] = sum;
    }
    if (THREADS_PER_VECTOR > 1) {
      tmp = sdata[threadIdx.x + 1];
      sum += tmp;
      sdata[threadIdx.x] = sum;
    }
    // First thread writes the result
    if (thread_lane == 0) {y[row] = sdata[threadIdx.x];}
  }
}

__global__ void kernel_spmv_CSR_vector_Mixed_Entrywise_Split(
    const unsigned int M,      // number of rows
    const float *AxS,          // float value iterator 1, matrix values
    float *xS,                 // float value iterator 2, dense vector
    const int *AjS,            // floats column iterator
    const int *ApS,            // floats row iterator
    const double *AxD,         // double value iterator 1, matrix values
    const double *xD,          // double value iterator 2, dense vector
    const int *AjD,            // doubles column iterator
    const int *ApD,            // doubles row iterator
    double *y,                 // value iterator 3, result vector
    const bool isAccumulative  // (T): y+=Ax, (F): y=Ax
) {
  const size_t VECTORS_PER_BLOCK = THREADS_PER_BLOCK / THREADS_PER_VECTOR;
  __shared__ volatile double
      sdata[VECTORS_PER_BLOCK * THREADS_PER_VECTOR + THREADS_PER_VECTOR / 2];  // padded to avoid reduction conditionals
  __shared__ volatile int ptrsS[VECTORS_PER_BLOCK][2];
  __shared__ volatile int ptrsD[VECTORS_PER_BLOCK][2];

  const int thread_id = THREADS_PER_BLOCK * blockIdx.x + threadIdx.x;  // global thread index
  const int thread_lane = threadIdx.x & (THREADS_PER_VECTOR - 1);      // thread index within the vector
  const int vector_id = thread_id / THREADS_PER_VECTOR;                // global vector index
  const int vector_lane = threadIdx.x / THREADS_PER_VECTOR;            // vector index within the block
  const int num_vectors = VECTORS_PER_BLOCK * gridDim.x;               // total number of active vectors

  for (int row = vector_id; row < M; row += num_vectors) {
    // use two threads to fetch Ap[row] and Ap[row+1]
    // this is considerably faster than the straightforward version
    if (thread_lane < 2) {
      ptrsS[vector_lane][thread_lane] = ApS[row + thread_lane];
      ptrsD[vector_lane][thread_lane] = ApD[row + thread_lane];
    }
    const int row_startS = ptrsS[vector_lane][0];  // same as: row_start = Ap[row];
    const int row_endS = ptrsS[vector_lane][1];    // same as: row_end   = Ap[row+1];
    const int row_startD = ptrsD[vector_lane][0];  // same as: row_start = Ap[row];
    const int row_endD = ptrsD[vector_lane][1];    // same as: row_end   = Ap[row+1];

    // initialize local sum
    double sum = 0.0;
    if (isAccumulative && (thread_lane == 0)) {
      sum = y[row];
    }

    // accumulate local sums
    // single precision
    if (THREADS_PER_VECTOR == 32 && row_endS - row_startS > 32) {
      // ensure aligned memory access to Aj and Ax
      int jj = row_startS - (row_startS & (THREADS_PER_VECTOR - 1)) + thread_lane;
      // accumulate local sums
      if (jj >= row_startS && jj < row_endS) sum += AxS[jj] * xS[AjS[jj]-1];
      // accumulate local sums
      for (jj += THREADS_PER_VECTOR; jj < row_endS; jj += THREADS_PER_VECTOR) sum += AxS[jj] * xS[AjS[jj]-1];
    } else {
      // accumulate local sums
      for (int jj = row_startS + thread_lane; jj < row_endS; jj += THREADS_PER_VECTOR) sum += AxS[jj] * xS[AjS[jj]-1];
    }

    // double precision
    if (THREADS_PER_VECTOR == 32 && row_endD - row_startD > 32) {
      // ensure aligned memory access to Aj and Ax
      int jj = row_startD - (row_startD & (THREADS_PER_VECTOR - 1)) + thread_lane;
      // accumulate local sums
      if (jj >= row_startD && jj < row_endD) sum += AxD[jj] * xD[AjD[jj]-1];
      // accumulate local sums
      for (jj += THREADS_PER_VECTOR; jj < row_endD; jj += THREADS_PER_VECTOR) sum += AxD[jj] * xD[AjD[jj]-1];
    } else {
      // accumulate local sums
      for (int jj = row_startD + thread_lane; jj < row_endD; jj += THREADS_PER_VECTOR) sum += AxD[jj] * xD[AjD[jj]-1];
    }

    // Store local sum in shared memory
    sdata[threadIdx.x] = sum;

    // Reduce local sums to row sum
    double tmp;
    if (THREADS_PER_VECTOR > 16) {
      tmp = sdata[threadIdx.x + 16];
      sum += tmp;
      sdata[threadIdx.x] = sum;
    }
    if (THREADS_PER_VECTOR > 8) {
      tmp = sdata[threadIdx.x + 8];
      sum += tmp;
      sdata[threadIdx.x] = sum;
    }
    if (THREADS_PER_VECTOR > 4) {
      tmp = sdata[threadIdx.x + 4];
      sum += tmp;
      sdata[threadIdx.x] = sum;
    }
    if (THREADS_PER_VECTOR > 2) {
      tmp = sdata[threadIdx.x + 2];
      sum += tmp;
      sdata[threadIdx.x] = sum;
    }
    if (THREADS_PER_VECTOR > 1) {
      tmp = sdata[threadIdx.x + 1];
      sum += tmp;
      sdata[threadIdx.x] = sum;
    }

    // First thread writes the result
    if (thread_lane == 0) {
      y[row] = sdata[threadIdx.x];
    }
  }
}

extern "C"  void entrywise_csr_(int *Anrowsc,
                                int *Annzc,
                                const int *Arows,
                                const int *Acols,
                                const double *Avals,
                                const double *xd,
                                double *b2,
                                int *Annzcs,
                                const int *ApS,
                                const int *AjS,
                                float *AxS,
                                float *xS)
{
  //定义A device
  int Anrows=*Anrowsc;
  int Annz=*Annzc;
  int AnnzS=*Annzcs;
  // double
  int *Arow_offset,*Aclo;
  double *Avalue,*xD,*y;
  // float
  int *Arow_offsetS,*AcloS;
  float *AvalueS,*xDS;

  //float
  hipMalloc(((void **)(&xDS)),Anrows* sizeof(float ));//
  hipMemcpy(xDS,xS,Anrows* sizeof(float ),hipMemcpyHostToDevice);

  hipMalloc(((void **)(&AvalueS)),AnnzS* sizeof(float ));//
  hipMemcpy(AvalueS,AxS,AnnzS* sizeof(float ),hipMemcpyHostToDevice);

  hipMalloc(((void **)(&Arow_offsetS)),(Anrows+1)* sizeof(int ));//
  hipMemcpy(Arow_offsetS,ApS,(Anrows+1)* sizeof(int ),hipMemcpyHostToDevice);

  hipMalloc(((void **)(&AcloS)),AnnzS* sizeof(int ));
  hipMemcpy(AcloS,AjS,AnnzS* sizeof(int ),hipMemcpyHostToDevice);//A.cols

  //double
  hipMalloc(((void **)(&xD)),Anrows* sizeof(double ));//
  hipMemcpy(xD,xd,Anrows* sizeof(double ),hipMemcpyHostToDevice);

  hipMalloc(((void **)(&Avalue)),Annz* sizeof(double ));//
  hipMemcpy(Avalue,Avals,Annz* sizeof(double ),hipMemcpyHostToDevice);

  hipMalloc(((void **)(&Arow_offset)),(Anrows+1)* sizeof(int ));//
  hipMemcpy(Arow_offset,Arows,(Anrows+1)* sizeof(int ),hipMemcpyHostToDevice);

  hipMalloc(((void **)(&Aclo)),Annz* sizeof(int ));
  hipMemcpy(Aclo,Acols,Annz* sizeof(int ),hipMemcpyHostToDevice);//A.cols

  hipMalloc(((void **)(&y)),Anrows* sizeof(double ));
  hipMemcpy(y,b2,Anrows* sizeof(double),hipMemcpyHostToDevice);//y

  const size_t VECTORS_PER_BLOCK  = THREADS_PER_BLOCK / THREADS_PER_VECTOR;//一个块中计算了多少行
  const size_t MAX_BLOCKS  = 2048;//cusp::system::cuda::detail::max_active_blocks
  const size_t NUM_BLOCKS = min(MAX_BLOCKS, (Anrows + (VECTORS_PER_BLOCK - 1)) / VECTORS_PER_BLOCK);
  
  // spmv_cusp<<< NUM_BLOCKS,THREADS_PER_BLOCK,0 >>>(Anrows,Avalue,Arow_offset,Aclo,xD,y);
  kernel_spmv_CSR_vector_Mixed_Entrywise_Split<<< NUM_BLOCKS,THREADS_PER_BLOCK,0 >>>(Anrows,AxS,xDS,AjS,ApS,Avalue,xD,Aclo,Arow_offset,y,false);
  hipDeviceSynchronize();
  hipMemcpy(b2,y,Anrows* sizeof(double ),hipMemcpyDeviceToHost);
  
  hipFree(Arow_offset);
  hipFree(Aclo);
  hipFree(Avalue);
  hipFree(xD);
  hipFree(y);

  hipFree(Arow_offsetS);
  hipFree(AcloS);
  hipFree(AvalueS);
  hipFree(xDS);
}

extern "C"  void xjf_csr_(int *Anrowsc,
                                int *Annzc,
                                const int *Arows,
                                const int *Acols,
                                const double *Avals,
                                const double *xd,
                                double *b2)
{
  //定义A device
  int Anrows=*Anrowsc;
  int Annz=*Annzc;
  int *Arow_offset,*Aclo;
  double *Avalue,*xD,*y;

  hipMalloc(((void **)(&xD)),Anrows* sizeof(double ));//
  hipMemcpy(xD,xd,Anrows* sizeof(double ),hipMemcpyHostToDevice);

  hipMalloc(((void **)(&Avalue)),Annz* sizeof(double ));//
  hipMemcpy(Avalue,Avals,Annz* sizeof(double ),hipMemcpyHostToDevice);

  hipMalloc(((void **)(&Arow_offset)),(Anrows+1)* sizeof(int ));//
  hipMemcpy(Arow_offset,Arows,(Anrows+1)* sizeof(int ),hipMemcpyHostToDevice);

  hipMalloc(((void **)(&Aclo)),Annz* sizeof(int ));
  hipMemcpy(Aclo,Acols,Annz* sizeof(int ),hipMemcpyHostToDevice);//A.cols

  hipMalloc(((void **)(&y)),Anrows* sizeof(double ));
  hipMemcpy(y,b2,Anrows* sizeof(double),hipMemcpyHostToDevice);//y

  const size_t VECTORS_PER_BLOCK  = THREADS_PER_BLOCK / THREADS_PER_VECTOR;//一个块中计算了多少行
  const size_t MAX_BLOCKS  = 2048;//cusp::system::cuda::detail::max_active_blocks
  const size_t NUM_BLOCKS = min(MAX_BLOCKS, (Anrows + (VECTORS_PER_BLOCK - 1)) / VECTORS_PER_BLOCK);
  
  spmv_cusp<<< NUM_BLOCKS,THREADS_PER_BLOCK,0 >>>(Anrows,Avalue,Arow_offset,Aclo,xD,y);
  hipDeviceSynchronize();
  hipMemcpy(b2,y,Anrows* sizeof(double ),hipMemcpyDeviceToHost);
  
  hipFree(Arow_offset);
  hipFree(Aclo);
  hipFree(Avalue);
  hipFree(xD);
  hipFree(y);
}



// TM1=TM1+U(:,:,k0+kk,l0+ll)*dtmp
__device__ __host__ void f2add(double *TM1,double *U,double dtmp,int norb,int k,int l){
    int stride2=norb*norb;
    for(int i=0;i<norb;i++){
      for(int j=0;j<norb;j++){
        TM1[i*norb+j]+=U[l*stride2*norb+k*stride2+i*norb+j]*dtmp;

      }
    }
    // for(int i=0;i<stride2;i++){
    //   TM1[i]=TM1[i]+U[i]*dtmp;
      
    // }
}
//TM2=TM2+U(:,k0+kk,l0+ll,:)*dtmp
__device__ __host__ void f23add(double *TM2,double *U,double dtmp,int norb,int k,int l){
    int stride2=norb*norb,strides=l*stride2+k*norb,stride3=stride2*norb;
    for(int i=0;i<norb;i++){
      for(int j=0;j<norb;j++){
        TM2[i*norb+j]=U[strides+i*stride3+j]*dtmp;
      }
    }
}
// GM1(i0+ii,:,j0+jj,:)=(TM1+TM2)*2.0d0
__device__ __host__ void f13add(double *TM1,double *TM2,double *GM1,int norb,int i,int j){
    int stride2=norb*norb,strides=j*stride2+i,stride3=stride2*norb;
    for(int t=0;t<norb;t++){
      for(int s=0;s<norb;s++){
        // GM1[strides+t*stride3+s*norb]=(TM1[t*norb+s]+TM2[t*norb+s])*2.0;
        GM1[strides+s*stride3+t*norb]=(TM1[s*norb+t]+TM2[s*norb+t])*2.0;
      }
    }
}

extern "C" void matgeny_(int *occ,int *tot,double *GM1,double *T,
                         double *D,int *NP,int *NP2,int *NP3,
                         int *group,double *P,double *U){
    int nsub=*NP;
    int nact=*NP2;
    int norb=*NP3;
    int ioffset=0;
    int ioffset1=0;
    int i0,ix,joffset,joffset1,j0,jx,l0,lx,loffset,loffset1,k0,kx;
    int koffset,koffset1;   
    int stride3=nact*nact*nact;
    double dtmp;
    double *TM1,*TM2,*TM3;
    // for(int i=0;i<norb;i++){
    //   for(int j=0;j<norb;j++){
    //     printf("%lf ",U[i*norb+j]);
    //   }
    // }
    TM1=(double*)malloc(sizeof(double)*norb*norb);
    TM2=(double*)malloc(sizeof(double)*norb*norb);
    TM3=(double*)malloc(sizeof(double)*norb*norb);
    for(int i=0;i<nsub;i++){
        for(int ii=0;ii<occ[i];ii++){
            i0=ioffset;
            ix=ioffset1;
            joffset=0;
            joffset1=0;
            for(int j=0;j<nsub;j++){
                for(int jj=0;jj<occ[j];jj++){
                    j0=joffset;
                    jx=joffset1;
                    memset(TM1, 0.0, norb*norb * sizeof(double));
                    memset(TM2, 0.0, norb*norb * sizeof(double));
                    memset(TM3, 0.0, norb*norb * sizeof(double));
                    koffset=0;
                    koffset1=0;
                    for(int k=0;k<nsub;k++){
                        for(int kk=0;kk<occ[k];kk++){
                            k0=koffset;
                            kx=koffset1;
                            loffset=0;
                            loffset1=0;
                            for(int l=0;l<nsub;l++){
                                if(group[i+j*8]==group[k+l*8]){
                                    // int ll;
                                    for(int ll=0;ll<occ[l];ll++){
                                        l0=loffset;
                                        lx=loffset1;
                                        dtmp=P[(jx+jj)*stride3+(lx+ll)*nact*nact+(kx+kk)*nact+(ix+ii)];
                                        // if (ii==0&&jj==0&&kk==0&&ll<10)
                                        // {
                                        //   printf("dtmp  %lf ",dtmp);
                                        // }
                                        // TM1=TM1+U(:,:,k0+kk,l0+ll)*dtmp
                                        f2add(TM1,U,dtmp,norb,(k0+kk),(l0+ll));
                                        if (ii==0&&jj==0&&kk==0&&ll==0)
                                        {
                                          // for(int u= 0; u < norb; u++)
                                          // {
                                          //   printf("TM1  %lf ",TM1[u]);
                                          // }
                                          // printf("dtmp  %lf ",dtmp);
                                        }
                                    }

                                }
                                if(group[8*i+k]==group[8*j+l]){
                                    for(int ll=0;ll<occ[l];ll++){
                                        l0=loffset;
                                        lx=loffset1;
                                        dtmp=P[(kx+kk)*stride3+(lx+ll)*nact+(jx+jj)*nact+(ix+ii)]+
                                             P[(kx+kk)*stride3+(jx+jj)*nact+(lx+ll)*nact+(ix+ii)];
                                        f23add(TM2,U,dtmp,norb,(k0+kk),(l0+ll));
                                        // TM2=TM2+U(:,k0+kk,l0+ll,:)*dtmp
                                        if (ii==0&&jj==0&&kk==0&&ll==0)
                                        {
                                          for(int u= 0; u < norb; u++)
                                          {
                                            printf("TM1  %lf ",TM2[u]);
                                          }
                                          // printf("dtmp  %lf ",dtmp);
                                        }
                                    }
                                }
                                loffset=loffset+tot[l];
                                loffset1=loffset1+occ[l];
                            }
                        }
                        koffset=koffset+tot[k];
                        koffset1=koffset1+occ[k];
                    }
                    // GM1(i0+ii,:,j0+jj,:)=(TM1+TM2)*2.0d0
                    f13add(TM1,TM2,GM1,norb,(i0+ii),(j0+jj));
                    if(ii==0&&jj==0){
                      for(int f=0;f<norb;f++){
                        printf("GM1  %lf ",GM1[f]);
                      }
                    }
                }
                joffset=joffset+tot[j];
                joffset1=joffset1+occ[j];
            }
        }
        ioffset=ioffset+tot[i];
        ioffset1=ioffset1+occ[i];
    }
    free(TM1);
    free(TM2);
    free(TM3);
}