#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#define Block_Size 256

extern "C" void helloworldcuda_ (){
    printf("Hello world CUDA Routine !!\n");
    int deviceCount;
    int dev;
    hipGetDeviceCount(&deviceCount);
    dev=0;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    printf("\nDevice %d: on %d \"%s\"\n", dev, deviceCount, deviceProp.name);
}
__global__ void kernel_aplus(double *a, double *b ,int N){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;//j
    if(tid<N){
        for(int i=0;i<N;i++){
            a[tid*N+i]=b[tid*N+i]+b[i*N+tid];
        }
    }
    // for(int i=0;i<norb;i++){
    //     for(int j=0;j<norb;j++){
    //         // a[j][i]=F[j][i]+F[i][j]
    //         a[j*N+i]=b[j*N+i]+b[i*N+j]
    //     }
    // }
}
extern "C" void aplus_c_(double *a,double *b,int *NP){   
    printf("Enter GPU aplus  \n");
    int N=*NP;
    int size=N*N;
    int threads_per_block=Block_Size;
    int num_blocks=(N+threads_per_block-1)/threads_per_block;
    double *a_d,*b_d;
    hipMalloc( (void **)&a_d, sizeof(double) * size );
    hipMalloc( (void **)&b_d, sizeof(double) * size );

    hipMemcpy( a_d, a, sizeof(double) * size, hipMemcpyHostToDevice );
    hipMemcpy( b_d, b, sizeof(double) * size, hipMemcpyHostToDevice );
    
    kernel_aplus<<<num_blocks,threads_per_block>>>(a_d,b_d,N);
    hipDeviceSynchronize();
    hipMemcpy( a, a_d, sizeof(double) * size, hipMemcpyDeviceToHost );
    
    hipFree(a_d);
    hipFree(b_d);
}
__global__ void kernel_comp_hess(double *aplus,double *hess,double *dh,double *Y,double *Hdiag,int norb){
    double adelt=0.0;
    double dij,drj,dis,drs;
    int r = blockIdx.x * blockDim.x + threadIdx.x;
    if(r<norb){
        for(int i=0;i<norb;i++){
            for(int s=0;s<norb;s++){
                for(int j=0;j<norb;j++){
                    adelt=0.0;
                    dij = (i == j) ? 1.0 : 0.0;
                    drj = (r == j) ? 1.0 : 0.0;
                    dis = (i == s) ? 1.0 : 0.0;
                    drs = (r == s) ? 1.0 : 0.0;

                    // adelt=aplus[r*norb+s]*dij-aplus[i*norb+s]*drj-aplus[r*norb+j]*dis+aplus[i*norb+j]*drs;   //! all
                    // hess[r*norb*norb*norb+i*norb*norb+s*norb+j]=hess[r*norb*norb*norb+i*norb*norb+s*norb+j]+dh[r*norb*norb*norb+s*norb*norb+i*norb+j]-dh[i*norb*norb*norb+s*norb*norb+r*norb+j]-dh[r*norb*norb*norb+j*norb*norb+i*norb+s]+dh[i*norb*norb*norb+j*norb*norb+r*norb+s]; // all
                    // hess[r*norb*norb*norb+i*norb*norb+s*norb+j]=hess[r*norb*norb*norb+i*norb*norb+s*norb+j]+Y[r*norb*norb*norb+i*norb*norb+s*norb+j]-Y[i*norb*norb*norb+r*norb*norb+s*norb+j]-Y[r*norb*norb*norb+i*norb*norb+j*norb+s]+Y[i*norb*norb*norb+r*norb*norb+j*norb+s];  // all
                    //hess[r*norb*norb*norb+i*norb*norb+s*norb+j]=hess[r*norb*norb*norb+i*norb*norb+s*norb+j]-0.5*adelt;
                    
                    adelt=aplus[s*norb+r]*dij-aplus[s*norb+i]*drj-aplus[j*norb+r]*dis+aplus[j*norb+i]*drs;   //! all
                    hess[j*norb*norb*norb+s*norb*norb+i*norb+r]=hess[j*norb*norb*norb+s*norb*norb+i*norb+r]+dh[j*norb*norb*norb+i*norb*norb+s*norb+r]-dh[j*norb*norb*norb+r*norb*norb+s*norb+i]-dh[s*norb*norb*norb+i*norb*norb+j*norb+r]+dh[s*norb*norb*norb+r*norb*norb+j*norb+i]; // all
                    hess[j*norb*norb*norb+s*norb*norb+i*norb+r]=hess[j*norb*norb*norb+s*norb*norb+i*norb+r]+Y[j*norb*norb*norb+s*norb*norb+i*norb+r]-Y[j*norb*norb*norb+s*norb*norb+r*norb+i]-Y[s*norb*norb*norb+j*norb*norb+i*norb+r]+Y[s*norb*norb*norb+j*norb*norb+r*norb+i];  // all
                    hess[j*norb*norb*norb+s*norb*norb+i*norb+r]=hess[j*norb*norb*norb+s*norb*norb+i*norb+r]-0.5*adelt;
                }
            }
            Hdiag[i*norb+r]=hess[i*norb*norb*norb+r*norb*norb+i*norb+r];
        }
    }
    // if(r==0){
    //     printf("\ntest!!! \n");
    // }

}
extern "C" void comp_hess_c_ (double *aplus,double *hess,double *dh,double *Y,double *Hdiag,int *NP){
    int norb=*NP;
    int size=norb*norb;
    int size2=size*size;
    double *aplus_d,*hess_d,*dh_d,*Y_d,*Hdiag_d;
    int threads_per_block=64;
    int num_blocks=(norb+threads_per_block-1)/threads_per_block;
    hipMalloc( (void **)&aplus_d, sizeof(double) * size );
    hipMalloc( (void **)&hess_d, sizeof(double) * size2 );
    hipMalloc( (void **)&dh_d, sizeof(double) * size2 );
    hipMalloc( (void **)&Y_d, sizeof(double) * size2 );
    hipMalloc( (void **)&Hdiag_d, sizeof(double) * size );

    hipMemcpy( aplus_d, aplus, sizeof(double) * size, hipMemcpyHostToDevice);
    // cudaMemcpy( hess_d, hess, sizeof(double) * size2, cudaMemcpyHostToDevice);
    hipMemcpy( dh_d, dh, sizeof(double) * size2, hipMemcpyHostToDevice);
    hipMemcpy( Y_d, Y, sizeof(double) * size2, hipMemcpyHostToDevice);
    // cudaMemcpy( Hdiag_d, Hdiag, sizeof(double) * size, cudaMemcpyHostToDevice);
    kernel_comp_hess<<<num_blocks,threads_per_block>>>(aplus_d,hess_d,dh_d,Y_d,Hdiag_d,norb);
    hipDeviceSynchronize();
    hipMemcpy( hess, hess_d, sizeof(double) * size2, hipMemcpyDeviceToHost );
    hipMemcpy( Hdiag, Hdiag_d, sizeof(double) * size, hipMemcpyDeviceToHost );
    hipFree(aplus_d);
    hipFree(hess_d);
    hipFree(dh_d);
    hipFree(Y_d);
    hipFree(Hdiag_d);
    // double adelt=0.0;
    // double dij,drj,dis,drs;
    // for(int r=0;r<norb;r++){
    //     for(int i=0;i<norb;i++){
    //         for(int s=0;s<norb;s++){
    //             for(int j=0;j<norb;j++){
    //                 adelt=0.0;
    //                 dij = (i == j) ? 1.0 : 0.0;
    //                 drj = (r == j) ? 1.0 : 0.0;
    //                 dis = (i == s) ? 1.0 : 0.0;
    //                 drs = (r == s) ? 1.0 : 0.0;
    //                 adelt=aplus[r*norb+s]*dij-aplus[i*norb+s]*drj-aplus[r*norb+j]*dis+aplus[i*norb+j]*drs;   //! all
    //                 hess[r*norb*norb*norb+i*norb*norb+s*norb+j]=hess[r*norb*norb*norb+i*norb*norb+s*norb+j]+dh[r*norb*norb*norb+s*norb*norb+i*norb+j]-dh[i*norb*norb*norb+s*norb*norb+r*norb+j]-dh[r*norb*norb*norb+j*norb*norb+i*norb+s]+dh[i*norb*norb*norb+j*norb*norb+r*norb+s]; // all
    //                 hess[r*norb*norb*norb+i*norb*norb+s*norb+j]=hess[r*norb*norb*norb+i*norb*norb+s*norb+j]+Y[r*norb*norb*norb+i*norb*norb+s*norb+j]-Y[i*norb*norb*norb+r*norb*norb+s*norb+j]-Y[r*norb*norb*norb+i*norb*norb+j*norb+s]+Y[i*norb*norb*norb+r*norb*norb+j*norb+s];  // all
    //                 hess[r*norb*norb*norb+i*norb*norb+s*norb+j]=hess[r*norb*norb*norb+i*norb*norb+s*norb+j]-0.5*adelt;
    //             }
    //         }
    //         Hdiag[r*norb+i]=hess[r*norb*norb*norb+i*norb*norb+r*norb+i];
    //     }
    // }
    // double adelt=0.0;
    // double dij,drj,dis,drs;
    // for(int r=0;r<norb;r++){
    //     for(int i=0;i<norb;i++){
    //         for(int s=0;s<norb;s++){
    //             for(int j=0;j<norb;j++){
    //                 adelt=0.0;
    //                 dij = (i == j) ? 1.0 : 0.0;
    //                 drj = (r == j) ? 1.0 : 0.0;
    //                 dis = (i == s) ? 1.0 : 0.0;
    //                 drs = (r == s) ? 1.0 : 0.0;
    //                 adelt=aplus[s*norb+r]*dij-aplus[s*norb+i]*drj-aplus[j*norb+r]*dis+aplus[j*norb+i]*drs;   //! all
    //                 hess[j*norb*norb*norb+s*norb*norb+i*norb+r]=hess[j*norb*norb*norb+s*norb*norb+i*norb+r]+dh[j*norb*norb*norb+i*norb*norb+s*norb+r]-dh[j*norb*norb*norb+r*norb*norb+s*norb+i]-dh[s*norb*norb*norb+i*norb*norb+j*norb+r]+dh[s*norb*norb*norb+r*norb*norb+j*norb+i]; // all
    //                 hess[j*norb*norb*norb+s*norb*norb+i*norb+r]=hess[j*norb*norb*norb+s*norb*norb+i*norb+r]+Y[j*norb*norb*norb+s*norb*norb+i*norb+r]-Y[j*norb*norb*norb+s*norb*norb+r*norb+i]-Y[s*norb*norb*norb+j*norb*norb+i*norb+r]+Y[s*norb*norb*norb+j*norb*norb+r*norb+i];  // all
    //                 hess[j*norb*norb*norb+s*norb*norb+i*norb+r]=hess[j*norb*norb*norb+s*norb*norb+i*norb+r]-0.5*adelt;
    //             }
    //         }
    //     }
    // }
}
__device__ __host__ void mat_const12(double *GM1,int index_x,int index_y,double* T,double Dnum,int norb){
    //T 是一个对称矩阵
    int index_xy=index_y*norb+index_x;
    int stride3=norb*norb*norb;
    int stride2=norb*norb;
    for(int i=0;i<norb;i++){
        for(int j=0;j<norb;j++){
            // int uu=j*stride3+i*stride2+index_xy;
            // printf("uu: %d,i: %d,j: %d\n",uu,i,j);
            GM1[j*stride3+i*stride2+index_xy]=T[j*norb+i]*Dnum*2.0;
            // GM1(i0+ii,j0+jj,:,:)=T*D(ix+ii,jx+jj)*2.0d0
            // printf("T[i*norb+j]:%lf  ,  T[i*norb+j]:%lf \n",T[i*norb+j],T[j*norb+i]);
        }
    }

}
__global__ void kernel_dhgen(int *occ,int *tot,double *GM1,double *T,double *D,int nsub,int nact,int norb){
    int i = blockIdx.x * blockDim.x + threadIdx.x;//j
    int ioffset=0;
    int ioffset1=0;
    int i0,i1,i2,ix,joffset,joffset1,j0,j1,j2,jx;
    if(i<nsub){
        for(int ii=0;ii<occ[i];ii++){
            i0=ioffset;
            i1=tot[i];
            i2=occ[i];
            ix=ioffset1;
            joffset=0;
            joffset1=0;
            for(int j=0;j<nsub;j++){
                for(int jj=0;jj<occ[j];jj++){
                    j0=joffset;
                    j1=tot[j];
                    j2=occ[j];
                    jx=joffset1;
                    // GM1(i0+ii,j0+jj,:,:)=T*D(ix+ii,jx+jj)*2.0d0
                    mat_const12(GM1,(i0+ii),(j0+jj),T,D[(jx+jj)*nact+(ix+ii)],norb);
                }
            joffset=joffset+tot[i];
            joffset1=joffset1+occ[i];
            }
        }
        ioffset=ioffset+tot[i];
        ioffset1=ioffset1+occ[i];
    }
}
extern "C" void matgendh_(int *occ,int *tot,double *GM1,double *T,double *D,int *NP,int *NP2,int *NP3){
    printf("Enter GPU dhgen  \n");
    int nsub=*NP;
    int nact=*NP2;
    int norb=*NP3;
    int stride4=norb*norb*norb*norb;
    int stride2=norb*norb;
    int ioffset=0;
    int ioffset1=0;
    int i0,i1,i2,ix,joffset,joffset1,j0,j1,j2,jx=0;
    int threads_per_block=Block_Size;
    int num_blocks=(nsub+threads_per_block-1)/threads_per_block;
    int *d_occ,*d_tot;
    double *d_GM1,*d_T,*d_D;
    hipMalloc( (void **)&d_occ, sizeof(int) * nsub );
    hipMalloc( (void **)&d_tot, sizeof(int) * nsub );
    hipMalloc( (void **)&d_GM1, sizeof(double) * stride4 );
    hipMalloc( (void **)&d_T, sizeof(double) * stride2 );
    hipMalloc( (void **)&d_D, sizeof(double) * nact*nact );
    hipMemcpy( d_occ, occ, sizeof(int) * nsub, hipMemcpyHostToDevice );
    hipMemcpy( d_tot, tot, sizeof(int) * nsub, hipMemcpyHostToDevice );
    hipMemcpy( d_GM1, GM1, sizeof(double) * stride4, hipMemcpyHostToDevice );
    hipMemcpy( d_T, T, sizeof(double) * stride2, hipMemcpyHostToDevice );
    hipMemcpy( d_D, D, sizeof(double) * nact*nact, hipMemcpyHostToDevice );
    kernel_dhgen<<<num_blocks,threads_per_block>>>(d_occ,d_tot,d_GM1,d_T,d_D,nsub,nact,norb);
    hipDeviceSynchronize();
    hipMemcpy( occ, d_occ, sizeof(int) * nsub, hipMemcpyDeviceToHost );
    hipMemcpy( tot, d_tot, sizeof(int) * nsub, hipMemcpyDeviceToHost );
    hipMemcpy( GM1, d_GM1, sizeof(double) * stride4, hipMemcpyDeviceToHost );
    hipMemcpy( T, d_T, sizeof(double) * stride2, hipMemcpyDeviceToHost );
    hipMemcpy( D, d_D, sizeof(double) * nact*nact, hipMemcpyDeviceToHost );
    hipFree(d_occ);
    hipFree(d_tot);
    hipFree(d_GM1);
    hipFree(d_T);
    hipFree(d_D);
}
extern "C" void matgendhcpu_(int *occ,int *tot,double *GM1,double *T,double *D,int *NP,int *NP2,int *NP3){
    int nsub=*NP;
    int nact=*NP2;
    int norb=*NP3;
    int ioffset=0;
    int ioffset1=0;
    int i0,i1,i2,ix,joffset,joffset1,j0,j1,j2,jx=0;
    for(int i=0;i<nsub;i++){
        for(int ii=0;ii<occ[i];ii++){
            i0=ioffset;
            i1=tot[i];
            i2=occ[i];
            ix=ioffset1;
            joffset=0;
            joffset1=0;
            for(int j=0;j<nsub;j++){
                for(int jj=0;jj<occ[j];jj++){
                    j0=joffset;
                    j1=tot[j];
                    j2=occ[j];
                    jx=joffset1;
                    // GM1(i0+ii,j0+jj,:,:)=T*D(ix+ii,jx+jj)*2.0d0
                    mat_const12(GM1,(i0+ii),(j0+jj),T,D[(jx+jj)*nact+(ix+ii)],norb);
                }
            joffset=joffset+tot[i];
            joffset1=joffset1+occ[i];
            }
        }
        ioffset=ioffset+tot[i];
        ioffset1=ioffset1+occ[i];
    }
}
void mat_TM1(double *TM1,double *U,double dtmp,int z,int q,int norb){
    int index_xy=z*norb*norb*norb+q*norb*norb;
    for(int i=0;i<norb;i++){
        for(int j=0;j<norb;j++){
            TM1[i*norb+j]+=U[index_xy+i*norb+j];
            // TM1=TM1+U(:,:,k0+kk,l0+ll)*dtmp
        }
    }
    
}
extern "C" void matgeny_(int *occ,int *tot,double *GM1,double *T,double *D,int *NP,int *NP2,
int *NP3,int *group,double *P,double *U){
    int nsub=*NP;
    int nact=*NP2;
    int norb=*NP3;
    int ioffset=0;
    int ioffset1=0;
    int i0,i1,i2,ix,joffset,joffset1,j0,j1,j2,jx,lx,l0,l1,l2,loffset,loffset1,kx,k0,k1,k2;
    int koffset,koffset1;   
    double dtmp;
    double *TM1,*TM2,*TM3;
    for(int i=0;i<nsub;i++){
        for(int ii=0;ii<occ[i];ii++){
            i0=ioffset;
            i1=tot[i];
            i2=occ[i];
            ix=ioffset1;
            joffset=0;
            joffset1=0;
            for(int j=0;j<nsub;j++){
                for(int jj=0;jj<occ[j];jj++){
                    j0=joffset;
                    j1=tot[j];
                    j2=occ[j];
                    jx=joffset1;

                    memset(TM1, 0, norb*norb * sizeof(double));
                    memset(TM2, 0, norb*norb * sizeof(double));
                    memset(TM3, 0, norb*norb * sizeof(double));

                    koffset=0;
                    koffset1=0;
                    for(int k=0;k<nsub;k++){
                        for(int kk=0;kk<occ[k];kk++){
                            k0=koffset;
                            k1=tot[k];
                            k2=occ[k];
                            kx=koffset1;

                            loffset=0;
                            loffset1=0;
                            for(int l=0;l<nsub;l++){
                                if(group[8*i+j]==group[8*k+l]){
                                    for(int ll=1;ll<occ[l];ll++){
                                        l0=loffset;
                                        l1=tot[l];
                                        l2=occ[l];
                                        lx=loffset1;
                                        dtmp=P[(ix+ii)*nact*nact*nact+(kx+kk)*nact*nact+(lx+ll)*nact+(jx+jj)];
                                        // TM1=TM1+U(:,:,k0+kk,l0+ll)*dtmp
                                    }
                                }
                                if(group[8*i+k]==group[8*j+l]){
                                    for(int ll=1;ll<occ[l];ll++){
                                        l0=loffset;
                                        l1=tot[l];
                                        l2=occ[l];
                                        lx=loffset1;
                                        dtmp=P[(ix+ii)*nact*nact*nact+(jx+jj)*nact*nact+(lx+ll)*nact+(kx+kk)]+
                                             P[(ix+ii)*nact*nact*nact+(lx+ll)*nact*nact+(jx+jj)*nact+(kx+kk)];
                                
                                        // TM2=TM2+U(:,k0+kk,l0+ll,:)*dtmp
                                    }
                                }
                                loffset=loffset+tot[l];
                                loffset1=loffset1+occ[l];
                            }
                        }
                        koffset=koffset+tot[k];
                        koffset1=koffset1+occ[k];
                    }
                    // GM1(i0+ii,:,j0+jj,:)=(TM1+TM2)*2.0d0
                    free(TM1);
                    free(TM2);
                    free(TM3);
                }
                joffset=joffset+tot[j];
                joffset1=joffset1+occ[j];
            }
        }
        ioffset=ioffset+tot[i];
        ioffset1=ioffset1+occ[i];
    }    
}
