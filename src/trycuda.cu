#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#define Block_Size 256
#define NUM_BLOCKS_MAX 2147483647
__global__ void stretch(double *a, 
                        double *b,
                        int N){
    int index= blockIdx.x * blockDim.x + threadIdx.x;
    int i,j,stride2=N*N;
    double temp1,temp2;
    if(index<stride2){
        i=index/N;
        j=index%N;
        temp1=b[index];
        temp2=b[j*N+i];
        a[index]=temp1+temp2;
    }
}
__global__ void kernel_update_hess_stretch(double *aplus_ck,int norb,
                                           double *hess_ck,double *dh_ck,double *Y_ck,
                                           double *Hdiag_c,int stride3,int stride2){
    int index= blockIdx.x * blockDim.x + threadIdx.x;
    double adelt=0.0;
    int dij,drj,dis,drs;
    double temp,temp1;
    // if(index==0){printf("\n674374\n");}
    if(index<norb*stride3){
        int r = index / stride3;
        int i = (index / stride2) % norb;
        int s = (index / norb) % norb;
        int j = index % norb;
        dij = (i == j) ? 1.0 : 0.0;
        drj = (r == j) ? 1.0 : 0.0;
        dis = (i == s) ? 1.0 : 0.0;
        drs = (r == s) ? 1.0 : 0.0;
        adelt=0.0;
        adelt=aplus_ck[r*norb+s]*dij-aplus_ck[i*norb+s]*drj-aplus_ck[r*norb+j]*dis+
                    aplus_ck[i*norb+j]*drs;
        
        temp=hess_ck[r*stride3+i*stride2+s*norb+j];
        temp=temp+dh_ck[r*stride3+s*stride2+i*norb+j]-
        dh_ck[i*stride3+s*stride2+r*norb+j]-dh_ck[r*stride3+j*stride2+i*norb+s]+
        dh_ck[i*stride3+j*stride2+r*norb+s];
        temp=temp+
                Y_ck[r*stride3+i*stride2+s*norb+j]-Y_ck[i*stride3+r*stride2+s*norb+j]-
                Y_ck[r*stride3+i*stride2+j*norb+s]+Y_ck[i*stride3+r*stride2+j*norb+s];

        hess_ck[r*stride3+i*stride2+s*norb+j]=temp-0.5*adelt;

        if(s==norb&&j==norb){
            Hdiag_c[r*norb+i]=hess_ck[r*stride3+i*stride2+r*norb+i];
        }
    }
}
extern "C" void hess3c_(double *aplus_c,double *F_c,int *NP,
                        double *hess_c,double *dh_c,double *Y_c,
                        double *Hdiag_c){
    printf("\nEnter hess3c_ C++\n");
    int norb=*NP;
    int stride4=norb*norb*norb*norb,stride3=norb*norb*norb,stride2=norb*norb;
    int threads_per_block=Block_Size,num_blocks=(stride2+threads_per_block-1)/threads_per_block;
    num_blocks=min(NUM_BLOCKS_MAX,num_blocks);
    printf("num_blocks\n  %d ",num_blocks);
    double *aplus_c_d,*F_c_d;
    hipMalloc( (void **)&aplus_c_d, sizeof(double) * stride2 );
    hipMalloc( (void **)&F_c_d, sizeof(double) * stride2 );
    hipMemcpy( aplus_c_d, aplus_c, sizeof(double) * stride2, hipMemcpyHostToDevice );
    hipMemcpy( F_c_d, F_c, sizeof(double) * stride2, hipMemcpyHostToDevice );
    stretch<<<num_blocks,threads_per_block>>>(aplus_c_d,F_c_d,norb);
    hipDeviceSynchronize();
    // for(int index=0;index<stride2;index++){
    //     int i=index/norb;
    //     int j=index%norb;
    //     aplus_c[index]=F_c[index]+F_c[j*norb+i];
    // }
    // for(int i=0;i<norb;i++){
    //     for(int j=0;j<norb;j++){
    //         aplus_c[j*norb+i]=F_c[j*norb+i]+F_c[i*norb+j];
    //     }
    // }
    
    double *hess_c_d,*dh_c_d,*Y_c_d,*Hdiag_c_d;
    hipMalloc( (void **)&hess_c_d, sizeof(double) * stride4 );
    hipMalloc( (void **)&dh_c_d, sizeof(double) * stride4 );
    hipMalloc( (void **)&Y_c_d, sizeof(double) * stride4 );
    hipMalloc( (void **)&Hdiag_c_d, sizeof(double) * stride2 );
    hipMemcpy( hess_c_d, hess_c, sizeof(double) * stride4, hipMemcpyHostToDevice );
    hipMemcpy( dh_c_d, dh_c, sizeof(double) * stride4, hipMemcpyHostToDevice );
    hipMemcpy( Y_c_d, Y_c, sizeof(double) * stride4, hipMemcpyHostToDevice );
    hipMemcpy( Hdiag_c_d, Hdiag_c, sizeof(double) * stride2, hipMemcpyHostToDevice );
    num_blocks=(stride4+threads_per_block-1)/threads_per_block;
    printf("num_block%d\n",num_blocks);
    kernel_update_hess_stretch<<<num_blocks,threads_per_block>>>(aplus_c_d,norb,
                                           hess_c_d,dh_c_d,Y_c_d,Hdiag_c_d,stride3,stride2);
    hipDeviceSynchronize();
    
    hipMemcpy( aplus_c, aplus_c_d, sizeof(double) * stride2, hipMemcpyDeviceToHost );
    hipMemcpy( hess_c, hess_c_d, sizeof(double) * stride4, hipMemcpyDeviceToHost );
    hipMemcpy( Hdiag_c, Hdiag_c_d, sizeof(double) * stride4, hipMemcpyDeviceToHost );

    hipFree(aplus_c_d);
    hipFree(F_c_d);
    hipFree(hess_c_d);
    hipFree(dh_c_d);
    hipFree(Y_c_d);
    hipFree(Hdiag_c_d);

    // double adelt=0.0;
    // int dij=0,drj=0,dis=0,drs=0;
    // for(int index=0;index<norb*stride3;index++){
    //     int r = index / stride3;
    //     int i = (index / stride2) % norb;
    //     int s = (index / norb) % norb;
    //     int j = index % norb;
    //     dij = (i == j) ? 1.0 : 0.0;
    //     drj = (r == j) ? 1.0 : 0.0;
    //     dis = (i == s) ? 1.0 : 0.0;
    //     drs = (r == s) ? 1.0 : 0.0;
    //     adelt=0.0;
    //     adelt=aplus_c[r*norb+s]*dij-aplus_c[i*norb+s]*drj-aplus_c[r*norb+j]*dis+
    //                 aplus_c[i*norb+j]*drs;
        
    //     hess_c[r*stride3+i*stride2+s*norb+j]=
    //     hess_c[r*stride3+i*stride2+s*norb+j]+dh_c[r*stride3+s*stride2+i*norb+j]-
    //     dh_c[i*stride3+s*stride2+r*norb+j]-dh_c[r*stride3+j*stride2+i*norb+s]+
    //     dh_c[i*stride3+j*stride2+r*norb+s];

    //     hess_c[r*stride3+i*stride2+s*norb+j]=hess_c[r*stride3+i*stride2+s*norb+j]+
    //     Y_c[r*stride3+i*stride2+s*norb+j]-Y_c[i*stride3+r*stride2+s*norb+j]-
    //     Y_c[r*stride3+i*stride2+j*norb+s]+Y_c[i*stride3+r*stride2+j*norb+s];

    //     hess_c[r*stride3+i*stride2+s*norb+j]-=0.5*adelt;

    //     if(s==norb&&j==norb){
    //         Hdiag_c[r*norb+i]=hess_c[r*stride3+i*stride2+r*norb+i];
    //     }

    // }
    // for(int r=0;r<norb;r++){
    //     for(int i=0;i<norb;i++){
    //         for(int s=0;s<norb;s++){
    //             for(int j=0;j<norb;j++){
    //                 adelt=0.0;
    //                 dij = (i == j) ? 1.0 : 0.0;
    //                 drj = (r == j) ? 1.0 : 0.0;
    //                 dis = (i == s) ? 1.0 : 0.0;
    //                 drs = (r == s) ? 1.0 : 0.0;
    //                 adelt=aplus_c[r*norb+s]*dij-aplus_c[i*norb+s]*drj-aplus_c[r*norb+j]*dis+
    //                 aplus_c[i*norb+j]*drs;

    //                 hess_c[r*stride3+i*stride2+s*norb+j]=
    //                 hess_c[r*stride3+i*stride2+s*norb+j]+dh_c[r*stride3+s*stride2+i*norb+j]-
    //                 dh_c[i*stride3+s*stride2+r*norb+j]-dh_c[r*stride3+j*stride2+i*norb+s]+
    //                 dh_c[i*stride3+j*stride2+r*norb+s];

    //                 hess_c[r*stride3+i*stride2+s*norb+j]=hess_c[r*stride3+i*stride2+s*norb+j]+
    //                 Y_c[r*stride3+i*stride2+s*norb+j]-Y_c[i*stride3+r*stride2+s*norb+j]-
    //                 Y_c[r*stride3+i*stride2+j*norb+s]+Y_c[i*stride3+r*stride2+j*norb+s];

    //                 hess_c[r*stride3+i*stride2+s*norb+j]-=0.5*adelt;
    //             }
    //         }
    //         Hdiag_c[r*norb+i]=hess_c[r*stride3+i*stride2+r*norb+i];
    //     }
    // }
    printf("End C++\n");
}

