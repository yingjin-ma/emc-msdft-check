#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <omp.h>
#include <sys/time.h>
#include <sys/stat.h>
#define THREADS_PER_BLOCK_AG 32
#define THREADS_PER_BLOCK_AG22 192
#define THREADS_PER_VECTOR 32

#define CUDA_CHECK_CALL(call)                                                                                     \
  {                                                                                                               \
    hipError_t cudaStatus = call;                                                                                \
    if (hipSuccess != cudaStatus)                                                                                \
      fprintf(stderr, "error CUDA RT call \"%s\" in line %d of file %s failed with  %s (%d).\n", #call, __LINE__, \
              __FILE__, hipGetErrorString(cudaStatus), cudaStatus);                                              \
  }


__global__ void computeA_1(int *occ,int noffset,int noffset1,
                           int moffset,int moffset1,int joffset,int joffset1,
                           int n,int m,int j,int norb,int nocc,
                           double *A,double *T,double *D){

    const size_t VECTORS_PER_BLOCK = THREADS_PER_BLOCK_AG / THREADS_PER_VECTOR; //6 最好整除
    __shared__ volatile double sdata[VECTORS_PER_BLOCK * THREADS_PER_VECTOR + THREADS_PER_VECTOR / 2];

    const int thread_id = THREADS_PER_BLOCK_AG * blockIdx.x + threadIdx.x;  // global thread index
    const int thread_lane = threadIdx.x & (THREADS_PER_VECTOR - 1);      // thread index within the vector
    const int vector_id = thread_id / THREADS_PER_VECTOR;                // global vector index

    // const int vector_lane = threadIdx.x / THREADS_PER_VECTOR;            // vector index within the block
    // const int num_vectors = VECTORS_PER_BLOCK * gridDim.x;               // total number of active vectors

    int n1 = vector_id / occ[m];//row
    int m1 = vector_id % occ[m];//col

    double sum=0.0;
    if(THREADS_PER_VECTOR == 32 && occ[j] > 32){
        int jj = 0 - (0 & (THREADS_PER_VECTOR - 1)) + thread_lane;
        if (jj >= 0 && jj < occ[j]) sum += 2.0*T[(n1+noffset)+(jj+joffset)*norb]*D[(m1+moffset1)+(jj+joffset1)*nocc];
    }else{
        for(int jj=thread_lane;jj<occ[j];jj+=THREADS_PER_VECTOR){
            sum+=2.0*T[(n1+noffset)+(jj+joffset)*norb]*D[(m1+moffset1)+(jj+joffset1)*nocc];
        }
    }
    sdata[threadIdx.x] = sum;
    double tmp;
    if (THREADS_PER_VECTOR > 16) {tmp = sdata[threadIdx.x + 16];sum += tmp;sdata[threadIdx.x] = sum;}
    if (THREADS_PER_VECTOR > 8) {tmp = sdata[threadIdx.x + 8];sum += tmp;sdata[threadIdx.x] = sum;}
    if (THREADS_PER_VECTOR > 4) {tmp = sdata[threadIdx.x + 4];sum += tmp;sdata[threadIdx.x] = sum;}
    if (THREADS_PER_VECTOR > 2) {tmp = sdata[threadIdx.x + 2];sum += tmp;sdata[threadIdx.x] = sum;}
    if (THREADS_PER_VECTOR > 1) {tmp = sdata[threadIdx.x + 1];sum += tmp;sdata[threadIdx.x] = sum;}

    A[(n1+noffset)+(m1+moffset)*norb]+=sum;
}

__global__ void computeA_11(int *total,int *occ,int noffset,int noffset1,
                           int moffset,int moffset1,int joffset,int joffset1,
                           int n,int m,int j,int norb,int nocc,
                           double *A,double *T,double *D){
    int index= blockIdx.x * blockDim.x + threadIdx.x;
    int n1=index/occ[m];
    int m1=index%occ[m];
    if(index<occ[m]*total[n]){
        // printf("~~~%lf",D[index]);
        for(int j1=0;j1<occ[j];j1++){
            A[(n1+noffset)+(m1+moffset)*norb]=A[(n1+noffset)+(m1+moffset)*norb]
            +2.0*T[(n1+noffset)+(j1+joffset)*norb]*D[(m1+moffset1)+(j1+joffset1)*nocc];
        }
    }
    // for(int n1=0;n1<orb_total[n];n1++){
    //     for(int m1=0;m1<orb_occ[m];m1++){
    //         for(int j1=0;j1<orb_occ[j];j1++){
    //             A[(n1+noffset)+(m1+moffset)*norb]=A[(n1+noffset)+(m1+moffset)*norb]
    //             +2.0*T[(n1+noffset)+(j1+joffset)*norb]*D[(m1+moffset1)+(j1+joffset1)*nocc];
    //         }
    //     }
    // }
}

__global__ void computeA_22(int *total,int *occ,int noffset,int noffset1,
                           int moffset,int moffset1,int joffset,int joffset1,int koffset,int koffset1,int loffset,int loffset1,
                           int n,int m,int j,int k,int l,int norb,int nocc,
                           double *A,double *p,double *U,int stride2norb,int stride2nocc,int stride3nocc,int stride3norb){
    int index= blockIdx.x * blockDim.x + threadIdx.x;
    int n1=index/occ[m];
    int m1=index%occ[m];
    // if(index==10){
    
    // }
    if(index<occ[m]*total[n]){
        printf("~~~%lf",p[index]);
        for(int j1=0;j1<occ[j];j1++){
            for(int k1=0;k1<occ[k];k1++){
                for(int l1=0;l1<occ[l];l1++){
                    A[(n1+noffset)+(m1+moffset)*norb]=A[(n1+noffset)+(m1+moffset)*norb]
                    +2.0*p[(m1+moffset1)+(k1+koffset1)*nocc+(l1+loffset1)*stride2nocc+stride3nocc*(j1+joffset1)]
                    *U[(n1+noffset)+norb*(j1+joffset)+(k1+koffset)*stride2norb+(l1+loffset)*stride3norb];
                }
            }
        }
    }

    // for(int n1=0;n1<orb_total[n];n1++){
    //     for(int m1=0;m1<orb_occ[m];m1++){
    //         for(int j1=0;j1<orb_occ[j];j1++){
    //             for(int k1=0;k1<orb_occ[k];k1++){
    //                 for(int l1=0;l1<orb_occ[l];l1++){
    //                     A[(n1+noffset)+(m1+moffset)*norb]=A[(n1+noffset)+(m1+moffset)*norb]
    //                     +2.0*p[(m1+moffset1)+(k1+koffset1)*nocc+(l1+loffset1)*stride2nocc+stride3nocc*(j1+joffset1)]
    //                     *U[(n1+noffset)+norb*(j1+joffset)+(k1+koffset)*stride2norb+(l1+loffset)*stride3norb];
    //                 }
    //             }
    //         }
    //     }
    // }
    
}

extern "C" void operator_(int *nsub,double* A,double *T,double *D,
                          int *norbA,int *noccD,
                          int *orb_total,int *orb_occ,
                          double *p,double *U,double *G){
    timeval start,end,s1,e1;
    double runtime=0.0,r1=0.0;
    gettimeofday(&start,NULL);
    int orbnsub=*nsub;
    int norb=*norbA;
    int nocc=*noccD;
    int noffset=0,noffset1=0,moffset,moffset1,joffset,joffset1;
    int koffset=0,koffset1=0,loffset=0,loffset1=0;
    int stride3nocc=nocc*nocc*nocc,stride2nocc=nocc*nocc;
    int stride3norb=norb*norb*norb,stride2norb=norb*norb;
    int *occ_d,*total_d;
    double *A_d,*T_d,*D_d;
    CUDA_CHECK_CALL(hipMalloc( (void **)&occ_d,sizeof(int)*orbnsub));
    CUDA_CHECK_CALL(hipMalloc( (void **)&total_d,sizeof(int)*orbnsub));
    CUDA_CHECK_CALL(hipMalloc( (void **)&A_d,sizeof(double)*stride2norb));
    CUDA_CHECK_CALL(hipMalloc( (void **)&T_d,sizeof(double)*stride2norb));
    CUDA_CHECK_CALL(hipMalloc( (void **)&D_d,sizeof(double)*stride2nocc));
    CUDA_CHECK_CALL(hipMemcpy( occ_d,orb_occ,sizeof(int)*orbnsub,hipMemcpyHostToDevice));
    CUDA_CHECK_CALL(hipMemcpy( total_d,orb_total,sizeof(int)*orbnsub,hipMemcpyHostToDevice));
    CUDA_CHECK_CALL(hipMemcpy( A_d,A,sizeof(double)*stride2norb,hipMemcpyHostToDevice));
    CUDA_CHECK_CALL(hipMemcpy( T_d,T,sizeof(double)*stride2norb,hipMemcpyHostToDevice));
    CUDA_CHECK_CALL(hipMemcpy( D_d,D,sizeof(double)*stride2nocc,hipMemcpyHostToDevice));
    for(int n=0;n<orbnsub;n++){
        moffset=0;
        moffset1=0;
        for(int m=0;m<orbnsub;m++){
            joffset=0;
            joffset1=0;
            for(int j=0;j<orbnsub;j++){
                // int block_num_AG=orb_total[n]*orb_occ[m];
                // computeA_1<<<block_num_AG,THREADS_PER_BLOCK_AG>>>(occ_d,noffset,noffset1,moffset,moffset1,joffset,joffset1,n,m,j,norb,nocc,A_d,T_d,D_d);
                int block_num_AG11=(orb_total[n]*orb_occ[m]+THREADS_PER_BLOCK_AG-1)/THREADS_PER_BLOCK_AG;
                computeA_11<<<block_num_AG11,THREADS_PER_BLOCK_AG>>>(total_d,occ_d,noffset,noffset1,moffset,moffset1,joffset,joffset1,n,m,j,norb,nocc,A_d,T_d,D_d);
                // for(int n1=0;n1<orb_total[n];n1++){
                //     for(int m1=0;m1<orb_occ[m];m1++){
                //         for(int j1=0;j1<orb_occ[j];j1++){
                //             A[(n1+noffset)+(m1+moffset)*norb]=A[(n1+noffset)+(m1+moffset)*norb]
                //             +2.0*T[(n1+noffset)+(j1+joffset)*norb]*D[(m1+moffset1)+(j1+joffset1)*nocc];
                //         }
                //     }
                // }
                hipDeviceSynchronize();
                joffset=joffset+orb_total[j];
                joffset1=joffset1+orb_occ[j];
            }
            moffset=moffset+orb_total[m];
            moffset1=moffset1+orb_occ[m];
        }
        noffset=noffset+orb_total[n];
        noffset1=noffset1+orb_occ[n];
    }
    // cudaMemcpy( orb_occ,occ_d,sizeof(double)*orbnsub,cudaMemcpyDeviceToHost);
    hipMemcpy( A,A_d,sizeof(double)*stride2norb,hipMemcpyDeviceToHost);
    // cudaMemcpy( T,T_d,sizeof(double)*stride2norb,cudaMemcpyDeviceToHost);
    // cudaMemcpy( D,D_d,sizeof(double)*stride2nocc,cudaMemcpyDeviceToHost);
    hipFree(occ_d);
    hipFree(total_d);
    hipFree(A_d);
    hipFree(T_d);
    hipFree(D_d);
    // double *p_d,*U_d;
    // CUDA_CHECK_CALL(cudaMalloc( (void **)&p_d,sizeof(double)*stride3nocc*nocc));
    // CUDA_CHECK_CALL(cudaMalloc( (void **)&U_d,sizeof(double)*stride3norb*norb));
    // CUDA_CHECK_CALL(cudaMemcpy(p_d,p,sizeof(double)*stride3nocc*nocc,cudaMemcpyHostToDevice));
    // CUDA_CHECK_CALL(cudaMemcpy(U_d,U,sizeof(double)*stride3norb*norb,cudaMemcpyHostToDevice));
    printf("***()*(*)(*&(*&*&^*&^^*&(*()*)&*(^*&^%&%&%$&$^%$%&(&))))");
    

    noffset=0;
    noffset1=0;
    moffset=0;moffset1=0;joffset=0;joffset1=0;
    for(int n=0;n<orbnsub;n++){
        moffset=0;
        moffset1=0;
        for(int m=0;m<orbnsub;m++){
            joffset=0;
            joffset1=0;
            for(int j=0;j<orbnsub;j++){
                koffset=0;
                koffset1=0;
                for(int k=0;k<orbnsub;k++){
                    loffset=0;
                    loffset1=0;
                    for(int l=0;l<orbnsub;l++){
                        // int block_num_AG22=(orb_total[n]*orb_occ[m]+THREADS_PER_BLOCK_AG-1)/THREADS_PER_BLOCK_AG;
                        // computeA_22<<<block_num_AG22,THREADS_PER_BLOCK_AG>>>(orb_total,orb_occ,noffset,noffset1,
                        //    moffset,moffset1,joffset,joffset1,koffset,koffset1,loffset,loffset1,
                        //    n,m,j,k,l,norb,nocc,
                        //    A_d,p_d,U_d,stride2norb,stride2nocc,stride3nocc,stride3norb);
                        // cudaDeviceSynchronize();
                        #pragma omp parallel for
                        for(int n1=0;n1<orb_total[n];n1++){
                            #pragma omp parallel for
                            for(int m1=0;m1<orb_occ[m];m1++){
                                for(int j1=0;j1<orb_occ[j];j1++){
                                    for(int k1=0;k1<orb_occ[k];k1++){
                                        for(int l1=0;l1<orb_occ[l];l1++){
                                            A[(n1+noffset)+(m1+moffset)*norb]=A[(n1+noffset)+(m1+moffset)*norb]
                                            +2.0*p[(m1+moffset1)+(k1+koffset1)*nocc+(l1+loffset1)*stride2nocc+stride3nocc*(j1+joffset1)]
                                            *U[(n1+noffset)+norb*(j1+joffset)+(k1+koffset)*stride2norb+(l1+loffset)*stride3norb];
                                        }
                                    }
                                }
                            }
                        }
                        loffset=loffset+orb_total[l];
                        loffset1=loffset1+orb_occ[l];
                    }
                    koffset=koffset+orb_total[k];
                    koffset1=koffset1+orb_occ[k];
                }
                joffset=joffset+orb_total[j];
                joffset1=joffset1+orb_occ[j];
            }
            moffset=moffset+orb_total[m];
            moffset1=moffset1+orb_occ[m];
        }
        noffset=noffset+orb_total[n];
        noffset1=noffset1+orb_occ[n];
    }
    // cudaMemcpy( A,A_d,sizeof(double)*stride2norb,cudaMemcpyDeviceToHost);
    hipFree(occ_d);
    hipFree(total_d);
    hipFree(A_d);
    // cudaFree(p_d);
    // cudaFree(U_d);

    // for(int i=0;i<stride2norb;i++){
    //     printf("~~%lf",A[i]);
    // }

    int ioffset=0;
    for(int i=0;i<orbnsub;i++){
        #pragma omp parallel for
        for(int k=0;k<orb_total[i];k++){
            for(int l=0;l<orb_total[i];l++){
                if(fabs(A[(k + ioffset)+(l+ioffset)*norb]) < 1.0e-9){
                    A[(k + ioffset)+(l+ioffset)*norb]=0.0;
                }
            }
        }
        ioffset=ioffset+orb_total[i];
    }

    #pragma omp parallel for
    for(int i=0;i<norb;i++){
        for(int j=0;j<norb;j++){
            if(fabs(A[i+j*norb])<1.0e-9){
                A[i+j*norb]=0.0;
            }
        }
    }
    
    // gettimeofday(&start,NULL);
    noffset=0;
    noffset1=0;
    for(int n=0;n<orbnsub;n++){
        moffset=0;
        moffset1=0;
        for(int m=0;m<orbnsub;m++){
            joffset=0;
            joffset1=0;
            for(int j=0;j<orbnsub;j++){
                koffset=0;
                koffset1=0;
                for(int k=0;k<orbnsub;k++){
                    #pragma omp parallel for
                    for(int n1=0;n1<orb_total[n];n1++){
                        #pragma omp parallel for
                        for(int m1=0;m1<orb_occ[m];m1++){
                            #pragma omp parallel for
                            for(int j1=0;j1<orb_occ[j];j1++){
                                #pragma omp parallel for
                                for(int k1=0;k1<orb_total[k];k1++){
                                    int mr=m1+moffset1;
                                    int jr=j1+joffset1;
                                    int ni=n1+noffset;
                                    int mi=m1+moffset;
                                    int ji=j1+joffset;
                                    int ki=k1+koffset;
                                    G[ni+mi*norb+ji*stride2norb+ki*stride3norb]+=2.0*T[ni+ki*norb]*D[mr+jr*nocc];
                                }
                            }
                        }
                    }
                    koffset=koffset+orb_total[k];
                    koffset1=koffset1+orb_occ[k];
                }
                joffset=joffset+orb_total[j];
                joffset1=joffset1+orb_occ[j];
            }
            moffset=moffset+orb_total[m];
            moffset1=moffset1+orb_occ[m];
        }
        noffset=noffset+orb_total[n];
        noffset1=noffset1+orb_occ[n];
    }
    hipDeviceSynchronize();
    gettimeofday(&end,NULL);
    runtime+=1e3*(end.tv_sec-start.tv_sec)+1e-3*(end.tv_usec-start.tv_usec);
    runtime/=1000;
    printf("G_1time:%lf s\n",runtime);


    gettimeofday(&s1,NULL);
    int kooffset=0,kooffset1=0;
    noffset=0;
    noffset1=0;
    for(int n=0;n<orbnsub;n++){
        moffset=0;
        moffset1=0;
        for(int m=0;m<orbnsub;m++){
            kooffset=0;
            kooffset1=0;
            for(int ko=0;ko<orbnsub;ko++){
                joffset=0;
                joffset1=0;
                for(int j=0;j<orbnsub;j++){
                    koffset=0;
                    koffset1=0;
                    for(int k=0;k<orbnsub;k++){
                        loffset=0;
                        loffset1=0;
                        for(int l=0;l<orbnsub;l++){
                            #pragma omp parallel for
                            for(int n1=0;n1<orb_total[n];n1++){
                                // #pragma omp parallel for
                                for(int m1=0;m1<orb_occ[m];m1++){
                                    for(int ko1=0;ko1<orb_occ[ko];ko1++){
                                        for(int j1=0;j1<orb_occ[j];j1++){
                                            for(int k1=0;k1<orb_occ[k];k1++){
                                                for(int l1=0;l1<orb_occ[l];l1++){
                                                    int mr=m1+moffset1;
                                                    int jr=j1+joffset1;
                                                    int kr=k1+koffset1;
                                                    int lr=l1+loffset1;
                                                    int ni=n1+noffset;
                                                    int mi=m1+moffset;
                                                    int koi=ko1+kooffset;
                                                    int ji=j1+joffset;
                                                    int ki=k1+koffset;
                                                    int li=l1+loffset;
                                                    G[ni+mi*norb+ji*stride2norb+koi*stride3norb]+=
                                                    2.0*p[mr+kr*nocc+lr*stride2nocc+jr*stride3nocc]
                                                    *U[ni+koi*norb+ki*stride2norb+li*stride3norb]
                                                    +2.0*2.0*p[mr+jr*nocc+lr*stride2nocc+kr*stride3nocc]
                                                    *U[ni+ki*norb+li*stride2norb+koi*stride3norb];
                                                    // printf("8 ");
                                                }
                                            }
                                        }
                                    }
                                }
                            }
                            loffset=loffset+orb_total[l];
                            loffset1=loffset1+orb_occ[l];
                        }
                        koffset=koffset+orb_total[k];
                        koffset1=koffset1+orb_occ[k];
                    }
                joffset=joffset+orb_total[j];
                joffset1=joffset1+orb_occ[j];   
                }
                kooffset=kooffset+orb_total[ko];
                kooffset1=kooffset1+orb_occ[ko];
            }
            moffset=moffset+orb_total[m];
            moffset1=moffset1+orb_occ[m];
        }
        noffset=noffset+orb_total[n];
        noffset1=noffset1+orb_occ[n];
    }

    hipDeviceSynchronize();
    gettimeofday(&e1,NULL);
    r1+=1e3*(e1.tv_sec-s1.tv_sec)+1e-3*(e1.tv_usec-s1.tv_usec);
    r1/=1000;
    printf("G_2time:%lf s\n",r1);
}



